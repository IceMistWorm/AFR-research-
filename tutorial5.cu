#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include "tutorial.h"
#include <optixu/optixu_aabb_namespace.h>

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float, number_of_parent_tiles, , ); //Stores the number of parent tiles. 


rtBuffer<unsigned int, 2>        rnd_seeds;

// add more buffers for drawing tiles
rtBuffer<int, 2>                 busy_buffer;
rtBuffer<int, 2>				 stencil_buffer;         //1 if should not be reprojected. 0 if it should.
rtBuffer<uchar4, 2>				 color_buffer;           //Stores the color value of the pixel before tranparency

rtBuffer<uint2, 1>				leaf_tile_indices;       //Stores the indices of the leaf tiles
rtBuffer<unsigned int, 1>		leaf_tile_sizes;         //Stores the size of the corresponding leaf tile

rtBuffer<uint2, 1>              parent_tile_indices;
rtBuffer<unsigned int, 1>		parent_tile_sizes;

rtBuffer<float, 1>              variance_buffer;         //Stores the variance
rtBuffer<float, 1>				parent_variance_buffer;  //Stores the variance of the parent tiles
rtBuffer<uint, 2>               pixel_at_tile_buffer;    //Store the pixel belongs to which tile

rtBuffer<float, 1>              render_elapse_time_buffer;
rtBuffer<int, 2>                mini_tile_buffer;
rtBuffer<int, 2>                random_map_buffer;

rtBuffer<float, 2>              tile_gradient_buffer;    //Stores gradient sum in the tile
rtBuffer<float3, 2>             extent_buffer;           //Stores extent

//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<int, 1>				 ray_per_sec_buffer;
rtBuffer<int, 1>				 raycounting_buffer;
rtBuffer<float3, 2>              float_color_buffer;
rtBuffer<float3, 2>              float_temp_buffer1;
rtBuffer<float3, 2>              float_temp_buffer2;
rtBuffer<float3, 2>              float_temp_buffer3;
rtBuffer<uchar4, 2>				 temp_buffer1;
rtBuffer<uchar4, 2>				 temp_buffer2;
rtBuffer<uchar4, 2>				 temp_buffer3;
rtBuffer<float3, 2>              crosshair_buffer, crosshair_buffer1, crosshair_buffer2, crosshair_buffer3;
rtBuffer<float, 2>               sample_time_buffer, sample_time_temp_buffer1, sample_time_temp_buffer2, sample_time_temp_buffer3;
rtBuffer<int, 1>                 screen_size_buffer;

rtBuffer<float3, 2>				 extent_buffer1;           //Stores extent
rtBuffer<float3, 2>				 extent_buffer2;           //Stores extent
rtBuffer<float3, 2>				 extent_buffer3;           //Stores extent
rtBuffer<float3, 1>              extent_at_tile;
rtBuffer<unsigned int, 1>       show_variance_buffer;
rtBuffer<unsigned int, 1>       show_gradient_buffer;
rtBuffer<unsigned int, 1>       show_tile_buffer;
rtBuffer<unsigned int, 1>       is_moving_buffer;

#define PI 3.1415926
// define weight matrix for Gaussian Filter (old, toooooo blurry)

__device__ float gaussian_1[1] = { 1.000000 };
__device__ float gaussian_3[3] = { 0.285472, 0.429056, 0.285472 };
__device__ float gaussian_5[5] = { 0.142483, 0.225851, 0.263331, 0.225851, 0.142483 };
__device__ float gaussian_7[7] = { 0.092665, 0.137812, 0.174867, 0.189313, 0.174867, 0.137812, 0.092665 };
__device__ float gaussian_9[9] = { 0.068208, 0.095616, 0.121705, 0.140662, 0.147616, 0.140662, 0.121705, 0.095616, 0.068208 };
__device__ float gaussian_11[11] = { 0.053809, 0.072025, 0.090358, 0.106246, 0.11709, 0.120945, 0.11709, 0.106246, 0.090358, 0.072025, 0.053809 };
__device__ float gaussian_13[13] = { 0.044386, 0.057305, 0.070626, 0.083094, 0.093325, 0.100059, 0.10241, 0.100059, 0.093325, 0.083094, 0.070626, 0.057305, 0.044386 };
__device__ float gaussian_15[15] = { 0.037739, 0.047357, 0.057387, 0.067154, 0.075886, 0.08281, 0.087265, 0.088802, 0.087265, 0.08281, 0.075886, 0.067154, 0.057387, 0.047357, 0.037739 };
__device__ float gaussian_17[17] = { 0.032818, 0.040246, 0.048031, 0.055783, 0.063047, 0.069345, 0.074226, 0.077317, 0.078376, 0.077317, 0.074226, 0.069345, 0.063047, 0.055783, 0.048031, 0.040246, 0.032818 };
__device__ float gaussian_19[19] = { 0.02902, 0.034926, 0.041127, 0.047385, 0.053419, 0.058922, 0.063592, 0.067153, 0.069384, 0.070144, 0.069384, 0.067153, 0.063592, 0.058922, 0.053419, 0.047385, 0.041127, 0.034926, 0.02902 };
__device__ float gaussian_21[21] = { 0.026011, 0.030815, 0.035862, 0.040996, 0.046037, 0.050784, 0.055029, 0.058575, 0.061247, 0.062908, 0.063472, 0.062908, 0.061247, 0.058575, 0.055029, 0.050784, 0.046037, 0.040996, 0.035862, 0.030815, 0.026011 };
__device__ float gaussian_23[23] = { 0.000093, 0.00033, 0.001039, 0.002903, 0.007184, 0.015758, 0.030627, 0.052751, 0.080517, 0.108910, 0.130549, 0.138679, 0.130549, 0.108910, 0.080517, 0.052751, 0.030627, 0.015758, 0.007184, 0.002903, 0.001039, 0.00033, 0.000093 };
__device__ float gaussian_25[25] = { 0.000044, 0.000159, 0.000511, 0.001470, 0.003785, 0.008718, 0.017966, 0.033126, 0.054647, 0.080660, 0.106520, 0.125863, 0.133062, 0.125863, 0.106520, 0.080660, 0.054647, 0.033126, 0.017966, 0.008718, 0.003785, 0.001470, 0.000511, 0.000159, 0.000044 };
__device__ float gaussian_27[27] = { 0.000021, 0.000077, 0.000251, 0.000740, 0.001970, 0.004731, 0.010250, 0.020031, 0.035313, 0.056153, 0.080548, 0.104224, 0.121650, 0.128084, 0.121650, 0.104224, 0.080548, 0.056153, 0.035313, 0.020031, 0.010250, 0.004731, 0.001970, 0.000740, 0.000251, 0.000077, 0.000021 };
__device__ float gaussian_29[29] = { 0.000010, 0.000037, 0.000123, 0.000371, 0.001016, 0.002530, 0.005723, 0.011760, 0.021952, 0.037225, 0.057344, 0.080249, 0.102022, 0.117826, 0.123621, 0.117826, 0.102022, 0.080249, 0.057344, 0.037225, 0.021952, 0.011760, 0.005723, 0.002530, 0.001016, 0.000371, 0.000123, 0.000037, 0.000010 };
__device__ float gaussian_31[31] = { 0.000005, 0.000018, 0.000060, 0.000185, 0.000520, 0.001337, 0.003140, 0.006740, 0.013226, 0.023722, 0.038889, 0.058274, 0.079816, 0.099924, 0.114345, 0.119600, 0.114345, 0.099924, 0.079816, 0.058274, 0.038889, 0.023722, 0.013226, 0.006740, 0.003140, 0.001337, 0.000520, 0.000185, 0.000060, 0.000018, 0.000005 };
__device__ float gaussian_33[33] = { 0.000002, 0.000009, 0.000029, 0.000092, 0.000265, 0.000700, 0.001699, 0.003790, 0.007770, 0.014640, 0.025350, 0.040339, 0.058992, 0.079283, 0.097922, 0.111148, 0.115943, 0.111148, 0.097922, 0.079283, 0.058992, 0.040339, 0.025350, 0.014640, 0.007770, 0.003790, 0.001699, 0.000700, 0.000265, 0.000092, 0.000029, 0.000009, 0.000002 };
//__device__ float gaussian_35[35] ={};
//__device__ float gaussian_37[37] = 
//__device__ float gaussian_39[39] = 
//__device__ float gaussian_41[41] = 

/*
// define weight matrix for Gaussian Filter (new!)

__device__ float gaussian_1[1] = { 1.000000 };
__device__ float gaussian_3[3] = { 0.282332, 0.435336, 0.282332};
__device__ float gaussian_5[5] = { 0.066576, 0.244739, 0.37737, 0.244739, 0.066576 };
__device__ float gaussian_7[7] = { 0.044533, 0.117512, 0.210298, 0.255313, 0.210298, 0.117512, 0.044533 };
__device__ float gaussian_9[9] = { 0.019538, 0.056663, 0.121206, 0.191261, 0.222666, 0.191261, 0.121206, 0.056663, 0.019538};
__device__ float gaussian_11[11] = { 0.00881, 0.027144, 0.065119, 0.121654, 0.176995, 0.200556, 0.176995, 0.121654, 0.065119, 0.027144, 0.00881};
__device__ float gaussian_13[13] = { 0.004046, 0.012996, 0.033758, 0.070923, 0.120521, 0.165661, 0.184193, 0.165661, 0.120521, 0.070923, 0.033758, 0.012996, 0.004046};
__device__ float gaussian_15[15] = { 0.001873, 0.006209, 0.017117, 0.039241, 0.074812, 0.118611, 0.156392, 0.171491, 0.156392, 0.118611, 0.074812,0.039241, 0.017117, 0.006209, 0.001873};
__device__ float gaussian_17[17] = { 0.000874, 0.00297, 0.008571, 0.021008, 0.043747, 0.077393, 0.116323, 0.148539, 0.161151, 0.148539, 0.116323, 0.077393, 0.043747, 0.021008, 0.008571, 0.00297, 0.000874};
__device__ float gaussian_19[19] = { 0.000414, 0.00143, 0.004271, 0.011027, 0.024604, 0.04744, 0.079054, 0.113849, 0.1417, 0.152423, 0.1417, 0.113849, 0.079054, 0.04744, 0.024604, 0.011027, 0.004271, 0.00143, 0.000414};
__device__ float gaussian_21[21] = { 0.000195, 0.000685, 0.002109, 0.005686, 0.01343, 0.027789, 0.05038, 0.080024, 0.111368, 0.135795, 0.145075, 0.135795, 0.111368, 0.080024, 0.05038, 0.027789, 0.01343, 0.005686, 0.002109, 0.000685,0.000195};
*/
/*

// test other gaussian filter weights sig = 1
__device__ float gaussian_1[1] = { 1.000000 };
__device__ float gaussian_3[3] = { 0.27901, 0.44198, 0.27901 };
__device__ float gaussian_5[5] = { 0.06136, 0.24477, 0.38774, 0.24477, 0.06136};
__device__ float gaussian_7[7] = { 0.00598, 0.060626,0.241843, 0.383103, 0.241843, 0.060626, 0.00598};
__device__ float gaussian_9[9] = { 0.000229, 0.005977, 0.060598, 0.241732, 0.382928, 0.241732, 0.060598, 0.005977, 0.000229};
__device__ float gaussian_11[11] = {0.000003, 0.000229, 0.005977, 0.060598, 0.24173, 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003};
__device__ float gaussian_13[13] = { 0, 0.000003, 0.000229, 0.005977, 0.060598, 0.24173, 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0};
__device__ float gaussian_15[15] = { 0, 0, 0.000003, 0.000229, 0.005977, 0.060598, 0.24173, 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0};
__device__ float gaussian_17[17] = { 0, 0, 0, 0.000003, 0.000229, 0.005977, 0.060598, 0.24173, 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0};
__device__ float gaussian_19[19] = { 0, 0, 0, 0, 0.000003, 0.000229, 0.005977, 0.060598, 0.24173, 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0};
__device__ float gaussian_21[21] = { 0, 0, 0, 0, 0, 0.000003, 0.000229, 0.005977, 0.060598, 0.24173, 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0};
*/

/*
// test other gaussian filter weights sig = 3
__device__ float gaussian_1[1] = { 1.000000 };
__device__ float gaussian_3[3] = { 0.327162, 0.345675, 0.327162};
__device__ float gaussian_5[5] = { 0.1784, 0.210431, 0.222338, 0.210431, 0.1784};
__device__ float gaussian_7[7] = { 0.106595, 0.140367, 0.165569, 0.174938, 0.165569, 0.140367, 0.106595};
__device__ float gaussian_9[9] = { 0.063327, 0.093095, 0.122589, 0.144599, 0.152781, 0.144599, 0.122589, 0.093095, 0.063327};
__device__ float gaussian_11[11] = { 0.035822, 0.05879, 0.086425, 0.113806, 0.13424, 0.141836, 0.13424, 0.113806, 0.086425, 0.05879, 0.035822};
__device__ float gaussian_13[13] = { 0.018816, 0.034474, 0.056577, 0.083173, 0.109523, 0.129188, 0.136498, 0.129188, 0.109523, 0.083173, 0.056577, 0.034474, 0.018816};
__device__ float gaussian_15[15] = { 0.009033, 0.018476, 0.033851, 0.055555, 0.08167, 0.107545, 0.126854, 0.134032, 0.126854, 0.107545, 0.08167, 0.055555, 0.033851, 0.018476, 0.009033};
__device__ float gaussian_17[17] = { 0.003924, 0.008962, 0.018331, 0.033585, 0.055119, 0.081029, 0.106701, 0.125858, 0.13298, 0.125858, 0.106701, 0.081029, 0.055119, 0.033585, 0.018331, 0.008962, 0.003924};
__device__ float gaussian_19[19] = { 0.001535, 0.003912, 0.008934, 0.018275, 0.033482, 0.05495, 0.08078, 0.106373, 0.125472, 0.132572, 0.125472, 0.106373, 0.08078, 0.05495, 0.033482, 0.018275, 0.008934, 0.003912, 0.001535};
__device__ float gaussian_21[21] = { 0.000539, 0.001533, 0.003908, 0.008925, 0.018255, 0.033446, 0.054891, 0.080693, 0.106259, 0.125337, 0.132429, 0.125337, 0.106259, 0.080693, 0.054891, 0.033446, 0.018255, 0.008925, 0.003908, 0.001533, 0.000539};


// test other gaussian filter weights sig = 7 (wayyyy to blurrrr)
__device__ float gaussian_1[1] = { 1.000000 };
__device__ float gaussian_3[3] = { 0.3322, 0.335601, 0.3322 };
__device__ float gaussian_5[5] = { 0.195938, 0.202018, 0.204087, 0.202018, 0.195938 };
__device__ float gaussian_7[7] = { 0.135679, 0.142769, 0.147199, 0.148706, 0.147199, 0.142769, 0.135679 };
__device__ float gaussian_9[9] = { 0.100856, 0.108311, 0.11397, 0.117507, 0.11871, 0.117507, 0.11397, 0.108311, 0.100856 };
__device__ float gaussian_11[11] = { 0.077718, 0.08518, 0.091475, 0.096255, 0.099242, 0.100258, 0.099242, 0.096255, 0.091475, 0.08518, 0.077718 };
__device__ float gaussian_13[13] = { 0.061002, 0.068236, 0.074787, 0.080315, 0.084512, 0.087134, 0.088026, 0.087134, 0.084512, 0.080315, 0.074787, 0.068236, 0.061002 };
__device__ float gaussian_15[15] = { 0.048277, 0.055112, 0.061647, 0.067566, 0.07256, 0.076352, 0.078721, 0.079527, 0.078721, 0.076352, 0.07256, 0.067566, 0.061647, 0.055112, 0.048277 };
__device__ float gaussian_17[17] = { 0.038265, 0.044582, 0.050895, 0.05693, 0.062396, 0.067007, 0.070509, 0.072697, 0.073441, 0.072697, 0.070509, 0.067007, 0.062396, 0.05693, 0.050895, 0.044582, 0.038265 };
__device__ float gaussian_19[19] = { 0.030234, 0.035951, 0.041886, 0.047817, 0.053487, 0.058623, 0.062955, 0.066245, 0.068301, 0.069, 0.068301, 0.066245, 0.062955, 0.058623, 0.053487, 0.047817, 0.041886, 0.035951, 0.030234 };
__device__ float gaussian_21[21] = { 0.023732, 0.028799, 0.034245, 0.039898, 0.045548, 0.050948, 0.05584, 0.059967, 0.063101, 0.065059, 0.065725, 0.065059, 0.063101, 0.059967, 0.05584, 0.050948, 0.045548, 0.039898, 0.034245, 0.028799, 0.023732 };
 */


 // temporal filter (sig = 1 )
 /*
 __device__ float gaussianT_1[1] = { 1.000000 };
 __device__ float gaussianT_3[2] = { 0.44198, 0.27901 };
 __device__ float gaussianT_5[3] = { 0.38774, 0.24477, 0.06136 };
 __device__ float gaussianT_7[4] = { 0.383103, 0.241843, 0.060626, 0.00598 };
 __device__ float gaussianT_9[5] = { 0.382928, 0.241732, 0.060598, 0.005977, 0.000229 };
 __device__ float gaussianT_11[6] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003 };
 __device__ float gaussianT_13[7] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0 };
 __device__ float gaussianT_15[8] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0 };
 __device__ float gaussianT_17[9] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0 };
 __device__ float gaussianT_19[10] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0 };
 __device__ float gaussianT_21[11] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_23[12] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_25[13] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_27[14] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_29[15] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_31[16] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_33[17] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 __device__ float gaussianT_35[18] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 __device__ float gaussianT_37[19] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 __device__ float gaussianT_39[20] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 __device__ float gaussianT_41[21] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_43[23] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 __device__ float gaussianT_45[24] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
 __device__ float gaussianT_47[25] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 __device__ float gaussianT_49[26] = { 0.382925, 0.24173, 0.060598, 0.005977, 0.000229, 0.000003, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 */

 // temporal filter (new!)

__device__ float gaussianT_1[1] = { 1.000000 };
__device__ float gaussianT_3[2] = { 0.435336, 0.282332 };
__device__ float gaussianT_5[3] = { 0.37737, 0.244739, 0.066576 };
__device__ float gaussianT_7[4] = { 0.255313, 0.210298, 0.117512, 0.044533 };
__device__ float gaussianT_9[5] = { 0.222666, 0.191261, 0.121206, 0.056663, 0.019538 };
__device__ float gaussianT_11[6] = { 0.200556, 0.176995, 0.121654, 0.065119, 0.027144, 0.00881 };
__device__ float gaussianT_13[7] = { 0.184193, 0.165661, 0.120521, 0.070923, 0.033758, 0.012996, 0.004046 };
__device__ float gaussianT_15[8] = { 0.171491, 0.156392, 0.118611, 0.074812,0.039241, 0.017117, 0.006209, 0.001873 };
__device__ float gaussianT_17[9] = { 0.161151, 0.148539, 0.116323, 0.077393, 0.043747, 0.021008, 0.008571, 0.00297, 0.000874 };
__device__ float gaussianT_19[10] = { 0.152423, 0.1417, 0.113849, 0.079054, 0.04744, 0.024604, 0.011027, 0.004271, 0.00143, 0.000414 };
__device__ float gaussianT_21[11] = { 0.145075, 0.135795, 0.111368, 0.080024, 0.050380, 0.027789, 0.01343, 0.005686, 0.002109, 0.000685, 0.000195 };
__device__ float gaussianT_23[12] = { 0.138679, 0.130549, 0.108910, 0.080517, 0.052751, 0.030627, 0.015758, 0.007184, 0.002903, 0.001039, 0.00033, 0.000093 };
__device__ float gaussianT_25[13] = { 0.133062, 0.125863, 0.106520, 0.080660, 0.054647, 0.033126, 0.017966, 0.008718, 0.003785, 0.001470, 0.000511, 0.000159, 0.000044 };
__device__ float gaussianT_27[14] = { 0.128084, 0.121650, 0.104224, 0.080548, 0.056153, 0.035313, 0.020031, 0.010250, 0.004731, 0.001970, 0.000740, 0.000251, 0.000077, 0.000021 };
__device__ float gaussianT_29[15] = { 0.123621, 0.117826, 0.102022, 0.080249, 0.057344, 0.037225, 0.021952, 0.011760, 0.005723, 0.002530, 0.001016, 0.000371, 0.000123, 0.000037, 0.000010 };
__device__ float gaussianT_31[16] = { 0.119600, 0.114345, 0.099924, 0.079816, 0.058274, 0.038889, 0.023722, 0.013226, 0.006740, 0.003140, 0.001337, 0.000520, 0.000185, 0.000060, 0.000018, 0.000005 };
__device__ float gaussianT_33[17] = { 0.115943, 0.111148, 0.097922, 0.079283, 0.058992, 0.040339, 0.025350, 0.014640, 0.007770, 0.003790, 0.001699, 0.000700, 0.000265, 0.000092, 0.000029, 0.000009, 0.000002 };
__device__ float gaussianT_35[18] = { 0.112606, 0.108209, 0.096020, 0.078679, 0.059533, 0.041596, 0.026838, 0.015989, 0.008797, 0.004469, 0.002096, 0.000908, 0.000363, 0.000134, 0.000046, 0.000014, 0.000004, 0.000001 };
__device__ float gaussianT_37[19] = { 0.109538, 0.105486, 0.094207, 0.078025, 0.059929, 0.042688, 0.028198, 0.017274, 0.009814, 0.005171, 0.002526, 0.001145, 0.000481, 0.000187, 0.000068, 0.000023, 0.000007, 0.000002, 0.000001 };
__device__ float gaussianT_39[20] = { 0.106711, 0.102961, 0.092484, 0.077336, 0.060204, 0.043631, 0.029437, 0.018489, 0.010811, 0.005885, 0.002982, 0.001407, 0.000618, 0.000253, 0.000096, 0.000034, 0.000011, 0.000003, 0.000001, 0 };
__device__ float gaussianT_41[21] = { 0.104090, 0.100606, 0.090840, 0.076624, 0.060379, 0.044447, 0.030566, 0.019636, 0.011785, 0.006607, 0.003460, 0.001693, 0.000774, 0.000330, 0.000132, 0.000049, 0.000017, 0.000006, 0.000002, 0, 0 };
__device__ float gaussianT_43[23] = { 0.101655, 0.098408, 0.089276, 0.075899, 0.060471, 0.045149, 0.031591, 0.020714, 0.012729, 0.007330, 0.003956, 0.002000, 0.000948, 0.000421, 0.000175, 0.000068, 0.000025, 0.000009, 0.000003, 0.000001, 0, 0 };
__device__ float gaussianT_45[24] = { 0.099381, 0.096345, 0.087781, 0.075167, 0.060492, 0.045753, 0.032523, 0.021728, 0.013642, 0.008050, 0.004464, 0.002327, 0.001140, 0.000525, 0.000227, 0.000092, 0.000035, 0.000013, 0.000004, 0.000001, 0, 0, 0 };
__device__ float gaussianT_47[25] = { 0.097255, 0.094408, 0.086356, 0.074434, 0.060455, 0.046269, 0.033368, 0.022676, 0.014521, 0.008762, 0.004982, 0.002669, 0.001348, 0.000641, 0.000287, 0.000121, 0.000048, 0.000018, 0.000006, 0.000002, 0.000001, 0, 0, 0 };
__device__ float gaussianT_49[26] = { 0.095258, 0.092581, 0.084992, 0.073702, 0.060369, 0.046708, 0.034135, 0.023564, 0.015366, 0.009464, 0.005506, 0.003026, 0.001571, 0.000770, 0.000357, 0.000156, 0.000064, 0.000025, 0.000009, 0.000003, 0.000001, 0, 0, 0, 0 };



// Generate random unsigned int in [0, 2^24)
static __host__ __device__ __inline__ unsigned int lcg(unsigned int &prev)
{
	const unsigned int LCG_A = 1664525u;
	const unsigned int LCG_C = 1013904223u;
	prev = (LCG_A * prev + LCG_C);
	return prev & 0x00FFFFFF;
}

// Generate random float in [0, 1)
static __host__ __device__ __inline__ float rnd(unsigned int &prev)
{
	return ((float)lcg(prev) / (float)0x01000000);
}


static __device__ __inline__ void frameless_rendering()
{
	size_t2 screen = output_buffer.size();

	volatile unsigned int seed = rnd_seeds[launch_index]; // volatile workaround for cuda 2.0 bug
	unsigned int new_seed = seed;
	float anytorandom = rnd(new_seed);
	float anytorandom2 = rnd(new_seed);
	rnd_seeds[launch_index] = new_seed;

	float x_offset = rnd(new_seed);
	float y_offset = rnd(new_seed);

	uint2 start_pixel;
	start_pixel.x = 0;
	start_pixel.y = 0;

	x_offset *= screen_size_buffer[0];
	y_offset *= screen_size_buffer[1];

	uint2 total_offset = make_uint2(x_offset, y_offset);
	uint2 rand_pixel = start_pixel + total_offset;

	//Ray creation
	float2 d = (make_float2(rand_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	if (busy_buffer[rand_pixel] == 0) //If pixel is free
	{
		atomicExch(&busy_buffer[rand_pixel], 1); //Set pixel to busy

		//output_buffer[rand_pixel] = make_color(prd.result);
		color_buffer[rand_pixel] = make_color(prd.result);
		stencil_buffer[rand_pixel] = 1;
		atomicExch(&busy_buffer[rand_pixel], 0); //Set pixel to free.
	}

}

// shoot 3 rays simultaneously, and calculate the gradient and add to the tile gradient buffer in this thread 
static __device__ __inline__ void shoot_3rays_new()
{
	// record the start time from rendering

	float start_to_render_time = render_elapse_time_buffer[0];
	clock_t shoot_ray_start_time = clock();

	size_t2 screen = output_buffer.size();

	volatile unsigned int seed = rnd_seeds[launch_index]; // volatile workaround for cuda 2.0 bug
	unsigned int new_seed = seed;
	float anytorandom = rnd(new_seed);
	float anytorandom2 = rnd(new_seed);
	rnd_seeds[launch_index] = new_seed;
	uint which_grid_calculate = launch_index.x % leaf_tile_indices.size();
	uint2 centre_pixel = make_uint2(leaf_tile_indices[which_grid_calculate].x, leaf_tile_indices[which_grid_calculate].y);

	//rtPrintf("launch index = %u  %u\n", launch_index.x, launch_index.y);

	for (int i = 0; i < 1; i++)
	{
		float x_offset = rnd(new_seed);
		float y_offset = rnd(new_seed);

		unsigned int tilesize = leaf_tile_sizes[launch_index.x];
		uint2 start_pixel;
		start_pixel.x = centre_pixel.x - tilesize / 2u;
		start_pixel.y = centre_pixel.y - tilesize / 2u;

		x_offset *= tilesize;
		y_offset *= tilesize;

		uint2 total_offset = make_uint2(x_offset, y_offset);
		uint2 rand_pixel = start_pixel + total_offset;
		float addrandomness = rnd_seeds[launch_index] * 0.000000000000039;

		/*
		if (busy_buffer[rand_pixel] == 1) // if pixel is busy
		{
		x_offset = rnd(new_seed);
		y_offset = rnd(new_seed);
		tilesize = leaf_tile_sizes[launch_index.x];
		start_pixel.x = centre_pixel.x - tilesize / 2u;
		start_pixel.y = centre_pixel.y - tilesize / 2u;

		x_offset *= tilesize;
		y_offset *= tilesize;
		total_offset = make_uint2(x_offset, y_offset);
		rand_pixel = start_pixel + total_offset;
		}*/


		//Right pixel
		uint2 right_pixel = rand_pixel + make_uint2(1.0f, 0.0f);
		//Top pixel
		uint2 top_pixel = rand_pixel + make_uint2(0.0f, 1.0f);


		//Ray creation (Center)
		float2 d = (make_float2(rand_pixel)) / make_float2(screen) * 2.f - 1.f;
		
		d.x += addrandomness;

		anytorandom = rnd(new_seed);
		anytorandom2 = rnd(new_seed);
		rnd_seeds[launch_index] = new_seed;
		addrandomness = rnd_seeds[launch_index] * 0.00000000000007;

		d.y += addrandomness;

		float3 ray_origin = eye;
		float3 ray_direction_center = normalize(d.x*U + d.y*V + W);
		
		anytorandom = rnd(new_seed);
		anytorandom2 = rnd(new_seed);
		rnd_seeds[launch_index] = new_seed;
		addrandomness = rnd_seeds[launch_index] * 0.00000000000007;


		optix::Ray ray = optix::make_Ray(ray_origin, ray_direction_center, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

		PerRayData_radiance prd;
		prd.importance = 1.f;
		prd.depth = 0;

		//Ray creation (Top)
		float2 d_top = (make_float2(top_pixel)) / make_float2(screen) * 2.f - 1.f;
		float3 ray_direction_top = normalize(d_top.x*U + d_top.y*V + W);
		
		d_top.x += addrandomness;

		anytorandom = rnd(new_seed);
		anytorandom2 = rnd(new_seed);
		rnd_seeds[launch_index] = new_seed;
		addrandomness = rnd_seeds[launch_index] * 0.00000000000007;

		d_top.y += addrandomness;

		anytorandom = rnd(new_seed);
		anytorandom2 = rnd(new_seed);
		rnd_seeds[launch_index] = new_seed;
		addrandomness = rnd_seeds[launch_index] * 0.00000000000007;

		optix::Ray ray_top = optix::make_Ray(ray_origin, ray_direction_top, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

		PerRayData_radiance prd_top;
		prd_top.importance = 1.f;
		prd_top.depth = 0;

		//Ray creation (Right)
		float2 d_right = (make_float2(right_pixel)) / make_float2(screen) * 2.f - 1.f;
		float3 ray_direction_right = normalize(d_right.x*U + d_right.y*V + W);
		
		d_right.x += addrandomness;

		anytorandom = rnd(new_seed);
		anytorandom2 = rnd(new_seed);
		rnd_seeds[launch_index] = new_seed;
		addrandomness = rnd_seeds[launch_index] * 0.00000000000007;

		d_right.y += addrandomness;

		anytorandom = rnd(new_seed);
		anytorandom2 = rnd(new_seed);
		rnd_seeds[launch_index] = new_seed;
		addrandomness = rnd_seeds[launch_index] * 0.00000000000007;
		
		optix::Ray ray_right = optix::make_Ray(ray_origin, ray_direction_right, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		

		PerRayData_radiance prd_right;
		prd_right.importance = 1.f;
		prd_right.depth = 0;


		rtTrace(top_object, ray, prd);
		rtTrace(top_object, ray_top, prd_top);
		rtTrace(top_object, ray_right, prd_right);

		float lum_centre = luminance(prd.result);
		float lum_right = luminance(prd_right.result);
		float lum_top = luminance(prd_top.result);

		uchar4 old_color_char = color_buffer[rand_pixel];
		float3 old_color = make_float3(old_color_char.z, old_color_char.y, old_color_char.x)*make_float3(1.0f / 255.99f);;

		//old_color = float_color_buffer[rand_pixel];

		float lum_temporal = luminance(old_color);

		clock_t shoot_ray_end_time = clock();
		float sample_elapse_time = (float)(((shoot_ray_end_time - shoot_ray_start_time) / 100000000.0f));

		//if (launch_index.x == 100u && launch_index.y == 100u)
		//	rtPrintf("srt = %f, set = %f\n", start_to_render_time, sample_elapse_time);

		float finish_sample_time_in_system = start_to_render_time + sample_elapse_time;

		finish_sample_time_in_system = start_to_render_time;


		float old_time = sample_time_buffer[rand_pixel];
		float new_time = finish_sample_time_in_system;
		float elapse_time = (new_time - old_time);

		//elapse_time /= 1000000000;


		if (elapse_time < 0)
			elapse_time *= -1;



		float YIQ_standard_center, YIQ_standard_right, YIQ_standard_top, YIQ_standard_old;

		YIQ_standard_center = prd.result.x * 0.299 + prd.result.y * 0.587 + prd.result.z * 0.114;
		YIQ_standard_right = prd_right.result.x * 0.299 + prd_right.result.y * 0.587 + prd_right.result.z * 0.114;
		YIQ_standard_top = prd_top.result.x * 0.299 + prd_top.result.y * 0.587 + prd_top.result.z * 0.114;
		YIQ_standard_old = old_color.x * 0.299 + old_color.y * 0.587 + old_color.z * 0.114;

		float_temp_buffer3[rand_pixel] = float_temp_buffer2[rand_pixel];
		float_temp_buffer2[rand_pixel] = float_temp_buffer1[rand_pixel];
		float_temp_buffer1[rand_pixel] = float_color_buffer[rand_pixel];
		float_color_buffer[rand_pixel] = prd.result;


		float g_x = lum_centre - lum_right;
		float g_y = lum_centre - lum_top;
		float g_t = lum_temporal - lum_centre;
		/*
		float g_x = YIQ_standard_center - YIQ_standard_right;
		float g_y = YIQ_standard_center - YIQ_standard_top;
		float g_t = YIQ_standard_old - YIQ_standard_center;
		*/

		if (g_x < 0)
			g_x *= -1;
		if (g_y < 0)
			g_y *= -1;
		if (g_t < 0)
			g_t *= -1;

		if (elapse_time == 0.0f) {
			elapse_time = 0.00001f;
			g_t = 0;
		}

		g_t /= elapse_time;
		//g_t *= exp(-3.27* elapse_time);


		//sample_time_buffer[rand_pixel] = new_time;

		// add crosshair to the crosshair_buffer

		float3 crosshair = make_float3(g_x, g_y, g_t);
		crosshair_buffer[rand_pixel] = crosshair;

		// calculate color and show gradient?
		uchar4 char_color = make_color(prd.result);
		float3 color = make_float3(char_color.z, char_color.y, char_color.x)*make_float3(1.0f / 255.99f);

		// extent per tile calculation 
		float3 extent;
		extent.x = extent.y = extent.z = 0.0f;
		//float vs = 0.04;
		//float rl = 100.0 / ((float(tilesize) * (float)tilesize) * (float)0.07);
		//vs = 1 / rl;
		float basic_tile_size = 64.0f;
		float vs = (ray_per_sec_buffer[0] / (256 * basic_tile_size))*(float(tilesize) * (float)tilesize) / basic_tile_size;
		//vs = (20 *(float(tilesize) * (float)tilesize) )/ basic_tile_size;;

		uint2 gradient_buffer_index_x = make_uint2(launch_index.x, 0.0f);
		uint2 gradient_buffer_index_y = make_uint2(launch_index.x, 1.0f);
		uint2 gradient_buffer_index_t = make_uint2(launch_index.x, 2.0f);
		float gradient_x_at_the_tile = tile_gradient_buffer[gradient_buffer_index_x];
		float gradient_y_at_the_tile = tile_gradient_buffer[gradient_buffer_index_y];
		float gradient_t_at_the_tile = tile_gradient_buffer[gradient_buffer_index_t];

		bool leaveLoop = false;
		while (!leaveLoop) {
			//if (busy_buffer[rand_pixel] == 0) //If pixel is free
			if (atomicExch(&(busy_buffer[rand_pixel]), 1u) == 0u) //If pixel is free
			{
				crosshair_buffer[rand_pixel] = crosshair;
				extent_buffer[rand_pixel] = extent;

				temp_buffer3[rand_pixel] = temp_buffer2[rand_pixel];
				temp_buffer2[rand_pixel] = temp_buffer1[rand_pixel];
				temp_buffer1[rand_pixel] = color_buffer[rand_pixel];

				color_buffer[rand_pixel] = make_color(prd.result);

				sample_time_temp_buffer3[rand_pixel] = sample_time_temp_buffer2[rand_pixel];
				sample_time_temp_buffer2[rand_pixel] = sample_time_temp_buffer1[rand_pixel];
				sample_time_temp_buffer1[rand_pixel] = sample_time_buffer[rand_pixel];
				sample_time_buffer[rand_pixel] = finish_sample_time_in_system;
				raycounting_buffer[0] += 1;
				leaveLoop = true;
				atomicExch(&busy_buffer[rand_pixel], 0); //Set pixel to free.
			}
			else {
				leaveLoop = true;
				//rayrejecting_buffer[0] += 1;
			}
		}
		leaveLoop = false;
		/**/
		while (!leaveLoop) {
			if (top_pixel.x < screen_size_buffer[0] && top_pixel.y < screen_size_buffer[1]) {
				if (atomicExch(&(busy_buffer[top_pixel]), 1u) == 0u) //If pixel is free
				{
					sample_time_temp_buffer3[top_pixel] = sample_time_temp_buffer2[top_pixel];
					sample_time_temp_buffer2[top_pixel] = sample_time_temp_buffer1[top_pixel];
					sample_time_temp_buffer1[top_pixel] = sample_time_buffer[top_pixel];
					color_buffer[top_pixel] = make_color(prd_top.result);

					sample_time_buffer[top_pixel] = finish_sample_time_in_system;
					temp_buffer3[top_pixel] = temp_buffer2[top_pixel];
					temp_buffer2[top_pixel] = temp_buffer1[top_pixel];
					temp_buffer1[top_pixel] = color_buffer[top_pixel];
					raycounting_buffer[0] += 1;
					leaveLoop = true;
					atomicExch(&busy_buffer[top_pixel], 0); //Set pixel to free.
				}
				else {
					leaveLoop = true;
					//rayrejecting_buffer[0] += 1;
				}
			}
			else {
				leaveLoop = true;
				//rayrejecting_buffer[0] += 1;
			}
		}

		leaveLoop = false;
		while (!leaveLoop) {
			if (right_pixel.x < screen_size_buffer[0] && right_pixel.y < screen_size_buffer[1]) {
				if (atomicExch(&(busy_buffer[right_pixel]), 1u) == 0u) //If pixel is free
				{
					temp_buffer3[right_pixel] = temp_buffer2[right_pixel];
					temp_buffer2[right_pixel] = temp_buffer1[right_pixel];
					temp_buffer1[right_pixel] = color_buffer[right_pixel];
					color_buffer[right_pixel] = make_color(prd_right.result);

					sample_time_temp_buffer3[right_pixel] = sample_time_temp_buffer2[right_pixel];
					sample_time_temp_buffer2[right_pixel] = sample_time_temp_buffer1[right_pixel];
					sample_time_temp_buffer1[right_pixel] = sample_time_buffer[right_pixel];
					sample_time_buffer[right_pixel] = finish_sample_time_in_system;
					raycounting_buffer[0] += 1;
					leaveLoop = true;
					atomicExch(&busy_buffer[right_pixel], 0); //Set pixel to free.
				}
				else {
					leaveLoop = true;
					//rayrejecting_buffer[0] += 1;
				}
			}
			else {
				leaveLoop = true;
				//rayrejecting_buffer[0] += 1;
			}
		}
	}
}


/*
shoot 3 rays simultaneously, and calculate the gradient and add to the tile gradient buffer in this thread
combine the random map method and our tiling
*/
static __device__ __inline__ void shoot_3rays_use_random_map_tile()
{
	// record the start time from rendering

	float start_to_render_time = render_elapse_time_buffer[0];
	clock_t shoot_ray_start_time = clock();

	size_t2 screen = output_buffer.size();

	// assign a thread to a certain tile by its launch index x, this will distribute threads equally to every tile

	uint which_grid_calculate = launch_index.x % leaf_tile_indices.size();
	uint2 centre_pixel = make_uint2(leaf_tile_indices[which_grid_calculate].x, leaf_tile_indices[which_grid_calculate].y);
	uint tilesize = leaf_tile_sizes[launch_index.x];

	uint count_mini_tile_quantity_length = tilesize / 8u;
	uint2 start_pixel;
	start_pixel.x = centre_pixel.x - tilesize / 2u;
	start_pixel.y = centre_pixel.y - tilesize / 2u;



	// start to pick a mini tile base on tile info, now randomly choose a mini-tile inside the tile

	uint2 start_mini_tile;
	start_mini_tile.x = start_pixel.x / 8u;
	start_mini_tile.y = start_pixel.y / 8u;

	uint2 offset_mini_tile;
	volatile unsigned int seed = rnd_seeds[launch_index]; // volatile workaround for cuda 2.0 bug
	unsigned int new_seed = seed;
	float anytorandom = rnd(new_seed);
	float anytorandom2 = rnd(new_seed);
	rnd_seeds[launch_index] = new_seed;

	offset_mini_tile.x = uint(rnd(new_seed) * 100) % count_mini_tile_quantity_length;
	offset_mini_tile.y = uint(rnd(new_seed) * 100) % count_mini_tile_quantity_length;

	uint2 which_mini_tile;

	which_mini_tile.x = start_mini_tile.x + offset_mini_tile.x;
	which_mini_tile.y = start_mini_tile.y + offset_mini_tile.y;

	if (which_mini_tile.x > 64u) {
		which_mini_tile.x = 0u;
	}
	if (which_mini_tile.y > 64u) {
		which_mini_tile.y = 0u;
	}

	start_pixel.x = which_mini_tile.x * 8u;
	start_pixel.y = which_mini_tile.y * 8u;

	uint mini_tile_pixel_lookup = mini_tile_buffer[which_mini_tile];
	mini_tile_buffer[which_mini_tile] = mini_tile_pixel_lookup + 1;
	if (mini_tile_buffer[which_mini_tile] > 64) {
		mini_tile_buffer[which_mini_tile] = 0u;
	}

	uint x_lookup = mini_tile_pixel_lookup / 8u;
	uint y_lookup = mini_tile_pixel_lookup % 8u;
	uint2 total_lookup = make_uint2(x_lookup, y_lookup);

	uint lookup_location = random_map_buffer[total_lookup];

	uint x_offset = lookup_location / 8u;
	uint y_offset = lookup_location % 8u;
	uint2 total_offset = make_uint2(x_offset, y_offset);
	uint2 rand_pixel = start_pixel + total_offset;


	//pixel_at_tile_buffer[rand_pixel] = which_grid_calculate;

	//Right pixel
	uint2 right_pixel = rand_pixel + make_uint2(1.0f, 0.0f);
	//Top pixel
	uint2 top_pixel = rand_pixel + make_uint2(0.0f, 1.0f);


	//Ray creation (Center)
	float2 d = (make_float2(rand_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction_center = normalize(d.x*U + d.y*V + W);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction_center, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	//Ray creation (Top)
	float2 d_top = (make_float2(top_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_direction_top = normalize(d_top.x*U + d_top.y*V + W);
	optix::Ray ray_top = optix::make_Ray(ray_origin, ray_direction_top, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd_top;
	prd_top.importance = 1.f;
	prd_top.depth = 0;

	//Ray creation (Right)
	float2 d_right = (make_float2(right_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_direction_right = normalize(d_right.x*U + d_right.y*V + W);
	optix::Ray ray_right = optix::make_Ray(ray_origin, ray_direction_right, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd_right;
	prd_right.importance = 1.f;
	prd_right.depth = 0;


	rtTrace(top_object, ray, prd);
	rtTrace(top_object, ray_top, prd_top);
	rtTrace(top_object, ray_right, prd_right);

	float lum_centre = luminance(prd.result);
	float lum_right = luminance(prd_right.result);
	float lum_top = luminance(prd_top.result);

	uchar4 old_color_char = color_buffer[rand_pixel];
	float3 old_color = make_float3(old_color_char.z, old_color_char.y, old_color_char.x)*make_float3(1.0f / 255.99f);;

	old_color = float_color_buffer[rand_pixel];

	float lum_temporal = luminance(old_color);

	clock_t shoot_ray_end_time = clock();
	float sample_elapse_time = (float)(((shoot_ray_end_time - shoot_ray_start_time) / 1000000000.0f));

	//if (launch_index.x == 100u && launch_index.y == 100u)
	//	rtPrintf("srt = %f, set = %f\n", start_to_render_time, sample_elapse_time);

	float finish_sample_time_in_system = start_to_render_time + sample_elapse_time;

	finish_sample_time_in_system = start_to_render_time;


	float old_time = sample_time_buffer[rand_pixel];
	float new_time = finish_sample_time_in_system;
	float elapse_time = (new_time - old_time);

	if (launch_index.x == 100u && launch_index.y == 100u) {
		//rtPrintf("el time = %f\n", elapse_time);
	}

	//elapse_time /= 1000000000;

	if (launch_index.x == 100u && launch_index.y == 100u)
		//rtPrintf("elapse_time = %f\n", elapse_time);


		if (elapse_time < 0) {
			//rtPrintf("nagative issue really here!!!!!!!!!");
			elapse_time *= -1;
		}



	float YIQ_standard_center, YIQ_standard_right, YIQ_standard_top, YIQ_standard_old;

	YIQ_standard_center = prd.result.x * 0.299 + prd.result.y * 0.587 + prd.result.z * 0.114;
	YIQ_standard_right = prd_right.result.x * 0.299 + prd_right.result.y * 0.587 + prd_right.result.z * 0.114;
	YIQ_standard_top = prd_top.result.x * 0.299 + prd_top.result.y * 0.587 + prd_top.result.z * 0.114;
	YIQ_standard_old = old_color.x * 0.299 + old_color.y * 0.587 + old_color.z * 0.114;

	float_temp_buffer3[rand_pixel] = float_temp_buffer2[rand_pixel];
	float_temp_buffer2[rand_pixel] = float_temp_buffer1[rand_pixel];
	float_temp_buffer1[rand_pixel] = float_color_buffer[rand_pixel];
	float_color_buffer[rand_pixel] = prd.result;


	float g_x = lum_centre - lum_right;
	float g_y = lum_centre - lum_top;
	float g_t = lum_temporal - lum_centre;
	/*
	float g_x = YIQ_standard_center - YIQ_standard_right;
	float g_y = YIQ_standard_center - YIQ_standard_top;
	float g_t = YIQ_standard_old - YIQ_standard_center;
	*/

	if (g_x < 0)
		g_x *= -1;
	if (g_y < 0)
		g_y *= -1;
	if (g_t < 0)
		g_t *= -1;


	if (elapse_time == 0.0f) {
		if (launch_index.x == 100u && launch_index.y == 100u) {
			//rtPrintf("old t = %f, start_to_render_time = %f,sample_elapse_time = %f, el time = %f\n", old_time,start_to_render_time, sample_elapse_time,elapse_time);
		}
		//g_t = crosshair_buffer[rand_pixel].z;
		g_t /= 0.000001f;
	}
	else {
		g_t /= elapse_time;
	}
	g_t /= 2;

	//g_t /= elapse_time;

	//g_t *= exp(-3.27* elapse_time);





	//sample_time_buffer[rand_pixel] = new_time;

	// add crosshair to the crosshair_buffer

	float3 crosshair = make_float3(g_x, g_y, g_t);


	// calculate color and show gradient?
	uchar4 char_color = make_color(prd.result);
	float3 color = make_float3(char_color.z, char_color.y, char_color.x)*make_float3(1.0f / 255.99f);

	// extent per tile calculation 
	float3 extent;
	extent.x = extent.y = extent.z = 0.0f;
	//float vs = 0.04;
	//float rl = 100.0 / ((float(tilesize) * (float)tilesize) * (float)0.07);
	//vs = 1 / rl;
	float basic_tile_size = 64.0f;
	float vs = (ray_per_sec_buffer[0] / (256 * basic_tile_size))*(float(tilesize) * (float)tilesize) / basic_tile_size;
	//vs = (20 *(float(tilesize) * (float)tilesize) )/ basic_tile_size;;

	uint2 gradient_buffer_index_x = make_uint2(launch_index.x, 0.0f);
	uint2 gradient_buffer_index_y = make_uint2(launch_index.x, 1.0f);
	uint2 gradient_buffer_index_t = make_uint2(launch_index.x, 2.0f);
	float gradient_x_at_the_tile = tile_gradient_buffer[gradient_buffer_index_x];
	float gradient_y_at_the_tile = tile_gradient_buffer[gradient_buffer_index_y];
	float gradient_t_at_the_tile = tile_gradient_buffer[gradient_buffer_index_t];

	bool leaveLoop = false;
	while (!leaveLoop) {
		//if (busy_buffer[rand_pixel] == 0) //If pixel is free
		if (atomicExch(&(busy_buffer[rand_pixel]), 1u) == 0u) //If pixel is free
		{
			crosshair_buffer[rand_pixel] = crosshair;
			extent_buffer[rand_pixel] = extent;

			temp_buffer3[rand_pixel] = temp_buffer2[rand_pixel];
			temp_buffer2[rand_pixel] = temp_buffer1[rand_pixel];
			temp_buffer1[rand_pixel] = color_buffer[rand_pixel];

			color_buffer[rand_pixel] = make_color(prd.result);

			sample_time_temp_buffer3[rand_pixel] = sample_time_temp_buffer2[rand_pixel];
			sample_time_temp_buffer2[rand_pixel] = sample_time_temp_buffer1[rand_pixel];
			sample_time_temp_buffer1[rand_pixel] = sample_time_buffer[rand_pixel];
			sample_time_buffer[rand_pixel] = finish_sample_time_in_system;
			raycounting_buffer[0] += 1;
			leaveLoop = true;
			atomicExch(&busy_buffer[rand_pixel], 0); //Set pixel to free.
		}
		else {
			leaveLoop = true;
			//rayrejecting_buffer[0] += 1;
		}
	}
	leaveLoop = false;
	/**/
	while (!leaveLoop) {
		if (atomicExch(&(busy_buffer[top_pixel]), 1u) == 0u) //If pixel is free
		{
			sample_time_temp_buffer3[top_pixel] = sample_time_temp_buffer2[top_pixel];
			sample_time_temp_buffer2[top_pixel] = sample_time_temp_buffer1[top_pixel];
			sample_time_temp_buffer1[top_pixel] = sample_time_buffer[top_pixel];
			color_buffer[top_pixel] = make_color(prd_top.result);

			sample_time_buffer[top_pixel] = finish_sample_time_in_system;
			temp_buffer3[top_pixel] = temp_buffer2[top_pixel];
			temp_buffer2[top_pixel] = temp_buffer1[top_pixel];
			temp_buffer1[top_pixel] = color_buffer[top_pixel];
			raycounting_buffer[0] += 1;
			leaveLoop = true;
			atomicExch(&busy_buffer[top_pixel], 0); //Set pixel to free.
		}
		else {
			leaveLoop = true;
			//rayrejecting_buffer[0] += 1;
		}
	}

	leaveLoop = false;
	while (!leaveLoop) {
		if (atomicExch(&(busy_buffer[right_pixel]), 1u) == 0u) //If pixel is free
		{
			temp_buffer3[right_pixel] = temp_buffer2[right_pixel];
			temp_buffer2[right_pixel] = temp_buffer1[right_pixel];
			temp_buffer1[right_pixel] = color_buffer[right_pixel];
			color_buffer[right_pixel] = make_color(prd_right.result);

			sample_time_temp_buffer3[right_pixel] = sample_time_temp_buffer2[right_pixel];
			sample_time_temp_buffer2[right_pixel] = sample_time_temp_buffer1[right_pixel];
			sample_time_temp_buffer1[right_pixel] = sample_time_buffer[right_pixel];
			sample_time_buffer[right_pixel] = finish_sample_time_in_system;
			raycounting_buffer[0] += 1;
			leaveLoop = true;
			atomicExch(&busy_buffer[right_pixel], 0); //Set pixel to free.
		}
		else {
			leaveLoop = true;
			//rayrejecting_buffer[0] += 1;
		}
	}

}



/*
shoot 3 rays simultaneously, and calculate the gradient and add to the tile gradient buffer in this thread
combine the random map method and our tiling
*/
static __device__ __inline__ void shoot_3rays_use_random_map_tile_test()
{
	// record the start time from rendering

	float start_to_render_time = render_elapse_time_buffer[0];
	clock_t shoot_ray_start_time = clock();

	size_t2 screen = output_buffer.size();

	// assign a thread to a certain tile by its launch index x, this will distribute threads equally to every tile

	uint which_grid_calculate = launch_index.x % leaf_tile_indices.size();
	uint2 centre_pixel = make_uint2(leaf_tile_indices[which_grid_calculate].x, leaf_tile_indices[which_grid_calculate].y);
	uint tilesize = leaf_tile_sizes[launch_index.x];

	uint count_mini_tile_quantity_length = tilesize / 8u;
	uint2 start_pixel;
	start_pixel.x = centre_pixel.x - tilesize / 2u;
	start_pixel.y = centre_pixel.y - tilesize / 2u;



	// start to pick a mini tile base on tile info, now randomly choose a mini-tile inside the tile

	uint2 start_mini_tile;
	start_mini_tile.x = start_pixel.x / 8u;
	start_mini_tile.y = start_pixel.y / 8u;

	uint2 offset_mini_tile;
	volatile unsigned int seed = rnd_seeds[launch_index]; // volatile workaround for cuda 2.0 bug
	unsigned int new_seed = seed;
	float anytorandom = rnd(new_seed);
	float anytorandom2 = rnd(new_seed);
	rnd_seeds[launch_index] = new_seed;

	offset_mini_tile.x = uint(rnd(new_seed) * 100) % count_mini_tile_quantity_length;
	offset_mini_tile.y = uint(rnd(new_seed) * 100) % count_mini_tile_quantity_length;

	uint2 which_mini_tile;

	which_mini_tile.x = start_mini_tile.x + offset_mini_tile.x;
	which_mini_tile.y = start_mini_tile.y + offset_mini_tile.y;

	if (which_mini_tile.x > 64u) {
		which_mini_tile.x %= 64u;
	}
	if (which_mini_tile.y > 64u) {
		which_mini_tile.y %= 64u;
	}

	start_pixel.x = which_mini_tile.x * 8u;
	start_pixel.y = which_mini_tile.y * 8u;

	uint mini_tile_pixel_lookup = mini_tile_buffer[which_mini_tile];
	mini_tile_buffer[which_mini_tile] = mini_tile_pixel_lookup + 1;
	if (mini_tile_buffer[which_mini_tile] > 64) {
		mini_tile_buffer[which_mini_tile] = 0u;
	}

	uint x_lookup = mini_tile_pixel_lookup / 8u;
	uint y_lookup = mini_tile_pixel_lookup % 8u;
	uint2 total_lookup = make_uint2(x_lookup, y_lookup);

	uint lookup_location = random_map_buffer[total_lookup];

	uint x_offset = lookup_location / 8u;
	uint y_offset = lookup_location % 8u;
	uint2 total_offset = make_uint2(x_offset, y_offset);
	uint2 rand_pixel = start_pixel + total_offset;


	//pixel_at_tile_buffer[rand_pixel] = which_grid_calculate;

	//Right pixel
	uint2 right_pixel = rand_pixel + make_uint2(1.0f, 0.0f);
	//Top pixel
	uint2 top_pixel = rand_pixel + make_uint2(0.0f, 1.0f);


	//Ray creation (Center)
	float2 d = (make_float2(rand_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction_center = normalize(d.x*U + d.y*V + W);

	optix::Ray ray = optix::make_Ray(ray_origin, ray_direction_center, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	//Ray creation (Top)
	float2 d_top = (make_float2(top_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_direction_top = normalize(d_top.x*U + d_top.y*V + W);
	optix::Ray ray_top = optix::make_Ray(ray_origin, ray_direction_top, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd_top;
	prd_top.importance = 1.f;
	prd_top.depth = 0;

	//Ray creation (Right)
	float2 d_right = (make_float2(right_pixel)) / make_float2(screen) * 2.f - 1.f;
	float3 ray_direction_right = normalize(d_right.x*U + d_right.y*V + W);
	optix::Ray ray_right = optix::make_Ray(ray_origin, ray_direction_right, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

	PerRayData_radiance prd_right;
	prd_right.importance = 1.f;
	prd_right.depth = 0;


	rtTrace(top_object, ray, prd);
	rtTrace(top_object, ray_top, prd_top);
	rtTrace(top_object, ray_right, prd_right);

	float lum_centre = luminance(prd.result);
	float lum_right = luminance(prd_right.result);
	float lum_top = luminance(prd_top.result);

	uchar4 old_color_char = color_buffer[rand_pixel];
	float3 old_color = make_float3(old_color_char.z, old_color_char.y, old_color_char.x)*make_float3(1.0f / 255.99f);;

	old_color = float_color_buffer[rand_pixel];

	float lum_temporal = luminance(old_color);

	clock_t shoot_ray_end_time = clock();
	float sample_elapse_time = (float)(((shoot_ray_end_time - shoot_ray_start_time) / 1000000000.0f));

	//if (launch_index.x == 100u && launch_index.y == 100u)
	//	rtPrintf("srt = %f, set = %f\n", start_to_render_time, sample_elapse_time);

	float finish_sample_time_in_system = start_to_render_time + sample_elapse_time;

	//finish_sample_time_in_system = start_to_render_time;


	float old_time = sample_time_buffer[rand_pixel];
	float new_time = finish_sample_time_in_system;
	float elapse_time = (new_time - old_time);

	if (launch_index.x == 100u && launch_index.y == 100u) {
		//rtPrintf("el time = %f\n", elapse_time);
	}

	//elapse_time /= 1000000000;

	if (launch_index.x == 100u && launch_index.y == 100u)
		//rtPrintf("elapse_time = %f\n", elapse_time);


		if (elapse_time < 0) {
			//rtPrintf("nagative issue really here!!!!!!!!!");
			elapse_time *= -1;
		}



	float YIQ_standard_center, YIQ_standard_right, YIQ_standard_top, YIQ_standard_old;

	YIQ_standard_center = prd.result.x * 0.299 + prd.result.y * 0.587 + prd.result.z * 0.114;
	YIQ_standard_right = prd_right.result.x * 0.299 + prd_right.result.y * 0.587 + prd_right.result.z * 0.114;
	YIQ_standard_top = prd_top.result.x * 0.299 + prd_top.result.y * 0.587 + prd_top.result.z * 0.114;
	YIQ_standard_old = old_color.x * 0.299 + old_color.y * 0.587 + old_color.z * 0.114;

	float_temp_buffer3[rand_pixel] = float_temp_buffer2[rand_pixel];
	float_temp_buffer2[rand_pixel] = float_temp_buffer1[rand_pixel];
	float_temp_buffer1[rand_pixel] = float_color_buffer[rand_pixel];
	float_color_buffer[rand_pixel] = prd.result;


	float g_x = lum_centre - lum_right;
	float g_y = lum_centre - lum_top;
	float g_t = lum_temporal - lum_centre;
	/*
	float g_x = YIQ_standard_center - YIQ_standard_right;
	float g_y = YIQ_standard_center - YIQ_standard_top;
	float g_t = YIQ_standard_old - YIQ_standard_center;
	*/

	if (g_x < 0)
		g_x *= -1;
	if (g_y < 0)
		g_y *= -1;
	if (g_t < 0)
		g_t *= -1;

	if (launch_index.x == 100u && launch_index.y == 100u) {
		//rtPrintf("tile size = %u\n", tilesize);
		//rtPrintf("old t = %f, start_to_render_time = %f,sample_elapse_time = %f, el time = %f\n", old_time, start_to_render_time, sample_elapse_time, elapse_time);
	}


	if (elapse_time == 0.0f) {
		
		//g_t = crosshair_buffer[rand_pixel].z;
		g_t /= 0.000001f;
		g_t = 0;
	}
	else {
		g_t /= elapse_time;
	}
	g_t /= 2;

	//g_t /= elapse_time;

	//g_t *= exp(-3.27* elapse_time);





	//sample_time_buffer[rand_pixel] = new_time;

	// add crosshair to the crosshair_buffer

	float3 crosshair = make_float3(g_x, g_y, g_t);


	// calculate color and show gradient?
	uchar4 char_color = make_color(prd.result);
	float3 color = make_float3(char_color.z, char_color.y, char_color.x)*make_float3(1.0f / 255.99f);

	// extent per tile calculation 
	float3 extent;
	extent.x = extent.y = extent.z = 0.0f;
	//float vs = 0.04;
	//float rl = 100.0 / ((float(tilesize) * (float)tilesize) * (float)0.07);
	//vs = 1 / rl;
	float basic_tile_size = 64.0f;
	float vs = (ray_per_sec_buffer[0] / (256 * basic_tile_size))*(float(tilesize) * (float)tilesize) / basic_tile_size;
	//vs = (20 *(float(tilesize) * (float)tilesize) )/ basic_tile_size;;

	uint2 gradient_buffer_index_x = make_uint2(launch_index.x, 0.0f);
	uint2 gradient_buffer_index_y = make_uint2(launch_index.x, 1.0f);
	uint2 gradient_buffer_index_t = make_uint2(launch_index.x, 2.0f);
	float gradient_x_at_the_tile = tile_gradient_buffer[gradient_buffer_index_x];
	float gradient_y_at_the_tile = tile_gradient_buffer[gradient_buffer_index_y];
	float gradient_t_at_the_tile = tile_gradient_buffer[gradient_buffer_index_t];

	if (launch_index.x == 100u && launch_index.y == 100u) {
		//rtPrintf("%f %f %f \n" , gradient_x_at_the_tile, gradient_y_at_the_tile, gradient_t_at_the_tile);
		//rtPrintf("%f %f %f \n", crosshair.x, crosshair.y, crosshair.z);
	}

	bool leaveLoop = false;
	while (!leaveLoop) {
		//if (busy_buffer[rand_pixel] == 0) //If pixel is free
		if (atomicExch(&(busy_buffer[rand_pixel]), 1u) == 0u) //If pixel is free
		{
			crosshair_buffer[rand_pixel] = crosshair;
			extent_buffer[rand_pixel] = extent;

			temp_buffer3[rand_pixel] = temp_buffer2[rand_pixel];
			temp_buffer2[rand_pixel] = temp_buffer1[rand_pixel];
			temp_buffer1[rand_pixel] = color_buffer[rand_pixel];

			color_buffer[rand_pixel] = make_color(prd.result);

			sample_time_temp_buffer3[rand_pixel] = sample_time_temp_buffer2[rand_pixel];
			sample_time_temp_buffer2[rand_pixel] = sample_time_temp_buffer1[rand_pixel];
			sample_time_temp_buffer1[rand_pixel] = sample_time_buffer[rand_pixel];
			sample_time_buffer[rand_pixel] = finish_sample_time_in_system;
			raycounting_buffer[0] += 1;
			leaveLoop = true;
			atomicExch(&busy_buffer[rand_pixel], 0); //Set pixel to free.
		}
		else {
			leaveLoop = true;
			//rayrejecting_buffer[0] += 1;
		}
	}
	leaveLoop = false;
	/**/
	while (!leaveLoop) {
		if (top_pixel.x < screen_size_buffer[0] && top_pixel.y < screen_size_buffer[0]) {
			if (atomicExch(&(busy_buffer[top_pixel]), 1u) == 0u) //If pixel is free
			{
				sample_time_temp_buffer3[top_pixel] = sample_time_temp_buffer2[top_pixel];
				sample_time_temp_buffer2[top_pixel] = sample_time_temp_buffer1[top_pixel];
				sample_time_temp_buffer1[top_pixel] = sample_time_buffer[top_pixel];
				color_buffer[top_pixel] = make_color(prd_top.result);

				sample_time_buffer[top_pixel] = finish_sample_time_in_system;
				temp_buffer3[top_pixel] = temp_buffer2[top_pixel];
				temp_buffer2[top_pixel] = temp_buffer1[top_pixel];
				temp_buffer1[top_pixel] = color_buffer[top_pixel];
				raycounting_buffer[0] += 1;
				leaveLoop = true;
				atomicExch(&busy_buffer[top_pixel], 0); //Set pixel to free.
			}
			else {
				leaveLoop = true;
				//rayrejecting_buffer[0] += 1;
			}
		}
		else{
			leaveLoop = true;
		//rayrejecting_buffer[0] += 1;
		}
	}

	leaveLoop = false;
	while (!leaveLoop) {
		if (right_pixel.x < screen_size_buffer[0] && right_pixel.y < screen_size_buffer[0]) {
			if (atomicExch(&(busy_buffer[right_pixel]), 1u) == 0u) //If pixel is free
			{
				temp_buffer3[right_pixel] = temp_buffer2[right_pixel];
				temp_buffer2[right_pixel] = temp_buffer1[right_pixel];
				temp_buffer1[right_pixel] = color_buffer[right_pixel];
				color_buffer[right_pixel] = make_color(prd_right.result);

				sample_time_temp_buffer3[right_pixel] = sample_time_temp_buffer2[right_pixel];
				sample_time_temp_buffer2[right_pixel] = sample_time_temp_buffer1[right_pixel];
				sample_time_temp_buffer1[right_pixel] = sample_time_buffer[right_pixel];
				sample_time_buffer[right_pixel] = finish_sample_time_in_system;
				raycounting_buffer[0] += 1;
				leaveLoop = true;
				atomicExch(&busy_buffer[right_pixel], 0); //Set pixel to free.
			}
			else {
				leaveLoop = true;
				//rayrejecting_buffer[0] += 1;
			}
		}
		else {
			leaveLoop = true;
			//rayrejecting_buffer[0] += 1;
		}
	}

}



static __device__ __inline__ void new_calculate_variance(bool leaf)
{
	// find the center pixel of a tile
	uint2 centre_pixel;
	unsigned int tile_size;
	if (leaf)
	{
		centre_pixel = make_uint2(leaf_tile_indices[launch_index.x].x, leaf_tile_indices[launch_index.x].y);
		tile_size = leaf_tile_sizes[launch_index.x];
	}
	else
	{
		centre_pixel = make_uint2(parent_tile_indices[launch_index.x].x, parent_tile_indices[launch_index.x].y);
		tile_size = parent_tile_sizes[launch_index.x];
	}
	//rtPrintf("tilesize = %u\n", tile_size);
	// find the start pixel of a tile (left to right, buttom to top)
	uint2 start_pixel;
	start_pixel.x = centre_pixel.x - tile_size / 2u;
	start_pixel.y = centre_pixel.y - tile_size / 2u;

	// find the end pixel of a tile
	uint2 end_pixel;
	end_pixel.x = centre_pixel.x + tile_size / 2u;
	end_pixel.y = centre_pixel.y + tile_size / 2u;

	// indicate the current pixel to iterate
	uint2 index_pixel = start_pixel;

	// initiate variables
	uchar4 mean_color;
	uchar4 char_color, char_color2, char_color3, char_color4;
	float3 color, color2, color3, color4;
	float3 color_square;
	float n = 0.0;
	float3 mean; mean.x = mean.y = mean.z = 0.0;
	float3 m2; m2.x = m2.y = m2.z = 0.0;
	float3 variance;
	float YIQ_standard_Y;


	float cross_hair_sum_x = 0.0f, cross_hair_sum_y = 0.0f, cross_hair_sum_t = 0.0f;
	float temp_sum_x = 0.0f, temp_sum_y = 0.0f, temp_sum_z = 0.0f;

	for (unsigned int i = 0; i < tile_size; i++)
	{
		for (unsigned int j = 0; j < tile_size; j++)
		{
			n++;
			//n = n + 4;
			uint2 offset = make_uint2(i, j);
			uint2 index = start_pixel + offset;

			char_color = color_buffer[index];
			char_color2 = temp_buffer1[index];
			char_color3 = temp_buffer2[index];
			char_color4 = temp_buffer3[index];
			color = make_float3(char_color.x, char_color.y, char_color.z)*make_float3(1.0f / 255.99f);
			color2 = make_float3(char_color2.x, char_color2.y, char_color2.z)*make_float3(1.0f / 255.99f);
			color3 = make_float3(char_color3.x, char_color3.y, char_color3.z)*make_float3(1.0f / 255.99f);
			color4 = make_float3(char_color4.x, char_color4.y, char_color4.z)*make_float3(1.0f / 255.99f);


			float3 delta = color - mean;

			mean += delta / n;
			m2 += delta * (color - mean);

			for (int k = 0; k < 4; k++) {
				if (k == 0) {
					float3 gradient = crosshair_buffer[index];
					cross_hair_sum_x += gradient.x;
					cross_hair_sum_y += gradient.y;
					cross_hair_sum_t += gradient.z;
				}
				else if (k == 1) {
					float3 gradient = crosshair_buffer1[index];
					gradient *= exp(-3.47);
					cross_hair_sum_x += gradient.x;
					cross_hair_sum_y += gradient.y;
					cross_hair_sum_t += gradient.z;
				}
				else if (k == 2) {
					float3 gradient = crosshair_buffer2[index];
					gradient *= exp(-3.47 * 2);
					cross_hair_sum_x += gradient.x;
					cross_hair_sum_y += gradient.y;
					cross_hair_sum_t += gradient.z;
				}
				else if (k == 3) {
					float3 gradient = crosshair_buffer3[index];
					gradient *= exp(-3.47 * 3);
					cross_hair_sum_x += gradient.x;
					cross_hair_sum_y += gradient.y;
					cross_hair_sum_t += gradient.z;
				}
			}
		}
	}

	variance = m2 / (n - 1);
	//rtPrintf("variance_show= %u\n", show_variance_buffer[launch_index.x]);

	float v = luminance(variance);
	YIQ_standard_Y = variance.x * 0.299 + variance.y * 0.587 + variance.z * 0.114;

	if (leaf)
	{
		//variance_buffer[launch_index.x] = v;
		variance_buffer[launch_index.x] = YIQ_standard_Y;

		uint2 gradient_buffer_index_x = make_uint2(launch_index.x, 0.0f);
		uint2 gradient_buffer_index_y = make_uint2(launch_index.x, 1.0f);
		uint2 gradient_buffer_index_t = make_uint2(launch_index.x, 2.0f);
		//rtPrintf(" %f %f \n", cross_hair_sum_x, cross_hair_sum_y);
		tile_gradient_buffer[gradient_buffer_index_x] = cross_hair_sum_x;
		tile_gradient_buffer[gradient_buffer_index_y] = cross_hair_sum_y;
		tile_gradient_buffer[gradient_buffer_index_t] = cross_hair_sum_t;
	}
	else
	{
		//parent_variance_buffer[launch_index.x] = v;
		parent_variance_buffer[launch_index.x] = YIQ_standard_Y;
	}

	if (show_variance_buffer[0] > 0) {
		for (unsigned int i = 0; i < tile_size; i++)
		{
			for (unsigned int j = 0; j < tile_size; j++)
			{
				uint2 offset = make_uint2(i, j);
				uint2 index = start_pixel + offset;
				char_color = color_buffer[index];
				color = make_float3(char_color.z, char_color.y, char_color.x)*make_float3(1.0f / 255.99f);
				YIQ_standard_Y = variance.x * 0.299 + variance.y * 0.587 + variance.z * 0.114;

				color.x = color.x + YIQ_standard_Y * 10;
				output_buffer[index] = make_color(color);
			}
		}
	}

}

static __device__ __inline__ void calculate_extent() {

	// find the center pixel of a tile
	uint2 centre_pixel;
	unsigned int tile_size = leaf_tile_sizes[launch_index.x];

	// find the start pixel of a tile (left to right, buttom to top)
	uint2 start_pixel;
	start_pixel.x = centre_pixel.x - tile_size / 2u;
	start_pixel.y = centre_pixel.y - tile_size / 2u;

	// indicate the current pixel to iterate
	uint2 index_pixel = start_pixel;

	// tempory change to gradient per pixel
	float basic_tile_size = 64.0f;
	float vs = (ray_per_sec_buffer[0] / (256 * basic_tile_size))*(float(tile_size) * (float)tile_size) / basic_tile_size;

	vs = 1/(((float)ray_per_sec_buffer[0] / 256.0f) / (float(tile_size) * (float)tile_size));

	//vs = ((float)ray_per_sec_buffer[0]/256.0f)();

	uint2 gradient_buffer_index_x = make_uint2(launch_index.x, 0.0f);
	uint2 gradient_buffer_index_y = make_uint2(launch_index.x, 1.0f);
	uint2 gradient_buffer_index_t = make_uint2(launch_index.x, 2.0f);

	float gradient_x_at_the_tile = tile_gradient_buffer[gradient_buffer_index_x];
	float gradient_y_at_the_tile = tile_gradient_buffer[gradient_buffer_index_y];
	float gradient_t_at_the_tile = tile_gradient_buffer[gradient_buffer_index_t];

	// some sanity check
	if (gradient_x_at_the_tile <= 0)
		gradient_x_at_the_tile *= -1;
	if (gradient_y_at_the_tile <= 0)
		gradient_y_at_the_tile *= -1;
	if (gradient_t_at_the_tile <= 0)
		gradient_t_at_the_tile *= -1;

	if (launch_index.x == 100u) {
		//rtPrintf("vs = %f, x = %f, y = %f, z = %f\n", vs, gradient_x_at_the_tile, gradient_y_at_the_tile, gradient_t_at_the_tile);
		//rtPrintf("tilesize = %u\n", tile_size);
	}


	float3 extent;

	float maxfiltersize = 100.0f, maxfiltersize_t = 25.0f;

	if (gradient_x_at_the_tile == 0.0f && gradient_y_at_the_tile == 0.0f && gradient_t_at_the_tile == 0.0f)
	{
		extent.x = extent.y = maxfiltersize;
		extent.z = maxfiltersize_t;
	}
	else if (gradient_x_at_the_tile == 0.0f && gradient_y_at_the_tile == 0.0f) {
		extent.x = maxfiltersize;
		extent.y = maxfiltersize;
		extent.z = 0.0f;
	}
	else if (gradient_y_at_the_tile == 0.0f && gradient_t_at_the_tile == 0.0f) {
		extent.x = 0.0f;
		extent.y = maxfiltersize;
		extent.z = maxfiltersize_t;
	}
	else if (gradient_x_at_the_tile == 0.0f && gradient_t_at_the_tile == 0.0f) {
		extent.x = maxfiltersize;
		extent.y = 0.0f;
		extent.z = maxfiltersize_t;
	}
	else if (gradient_x_at_the_tile == 0.0f) {
		extent.x = maxfiltersize;
		extent.y = 0.0f;
		extent.z = 0.0f;
	}
	else if (gradient_y_at_the_tile == 0.0f) {
		extent.x = 0.0f;
		extent.y = maxfiltersize;
		extent.z = 0.0f;
	}
	else if (gradient_t_at_the_tile == 0.0f) {
		extent.x = 0.0f;
		extent.y = 0.0f;
		extent.z = maxfiltersize_t;
	}
	else
	{

		extent.x = pow(((gradient_y_at_the_tile * gradient_t_at_the_tile * vs) / (gradient_x_at_the_tile * gradient_x_at_the_tile)), 0.33f);
		extent.y = pow(((gradient_x_at_the_tile * gradient_t_at_the_tile * vs) / (gradient_y_at_the_tile * gradient_y_at_the_tile)), 0.33f);
		extent.z = pow(((gradient_x_at_the_tile * gradient_y_at_the_tile * vs) / (gradient_t_at_the_tile * gradient_t_at_the_tile)), 0.33f);
	}
	extent_at_tile[launch_index.x] = extent;

	if (launch_index.x == 133u) {
		
		//rtPrintf(" In calculate extent: x = %f, y = %f, z = %f\n", extent.x, extent.y, extent.z);
		
		//rtPrintf("ray_per_sec_buffer = %d \n", ray_per_sec_buffer[0]);
		//rtPrintf("vs = %f, x = %f, y = %f, z = %f\n",vs, gradient_x_at_the_tile, gradient_y_at_the_tile, gradient_t_at_the_tile);
		//rtPrintf("screen_size_buffer0 = %d screen_size_buffer1 = %d\n", screen_size_buffer[0], screen_size_buffer[1]);
	}
}




static  __device__ __inline__ void no_reconstruction() {
	uint2 index = make_uint2(launch_index.x, launch_index.y);
	uchar4 final_color = color_buffer[index];
	output_buffer[index] = final_color;
}


static __device__ __inline__ void gaussian_filter_to_whole_image_gather()
{
	uint2 index = make_uint2(launch_index.x, launch_index.y);
	uchar4 final_color1 = color_buffer[index];
	uint which_grid_calculate = pixel_at_tile_buffer[index];
	
	if (which_grid_calculate > 256) {
		// assign a tile if the call is illegal, should not happen after every pixel is updated at least once
		//rtPrintf("not reach here\n, which_grid_calculate = %u", which_grid_calculate);

		which_grid_calculate = launch_index.x;
	}
	uint2 centre_pixel = make_uint2(leaf_tile_indices[which_grid_calculate].x, leaf_tile_indices[which_grid_calculate].y);
	unsigned int tilesize = leaf_tile_sizes[which_grid_calculate];

	float sig = 10.0f;
	float3 extents = extent_buffer[index];
	uint gaussian_filter_size = 21u;
	/*
	if(extents.y!=0)
	rtPrintf("search for y = %u \n", extents.y);
	*/
	float total_weight = 0.0f;
	uchar4 total_color, final_color;
	float3 tot_color;
	tot_color.x = tot_color.y = tot_color.z = 0.0f;

	uint gaussian_start_index_x = launch_index.x - gaussian_filter_size / 2 + 1;
	uint gaussian_start_index_y = launch_index.y - gaussian_filter_size / 2 + 1;
	uint2 guassian_start_index = make_uint2(gaussian_start_index_x, gaussian_start_index_y);

	for (unsigned int i = 0u; i < gaussian_filter_size; i++)
	{
		for (unsigned int j = 0u; j < gaussian_filter_size; j++)
		{
			float N = (1.0f) / (2.0f*PI*sig*sig);
			int x = (int)i;
			int y = (int)j;
			float E = exp(((x*x + y * y) / (2.0f*sig*sig))* (-1.0f));
			N *= E;
			uint color_index_x = guassian_start_index.x + i;
			uint color_index_y = guassian_start_index.y + j;

			if (color_index_x < 0 || color_index_x > screen_size_buffer[0] || color_index_y < 0 || color_index_y > screen_size_buffer[1]) {
				continue;
			}
			uint2 color_index = make_uint2(color_index_x, color_index_y);
			uchar4 color_info = color_buffer[color_index];
			float3 color_info_float = make_float3(color_info.z, color_info.y, color_info.x)*make_float3(1.0f / 255.99f);
			color_info_float *= make_float3(N);
			tot_color += color_info_float;
			total_weight += N;
		}
	}//for loop ends

	tot_color *= make_float3(1.0f / total_weight);
	final_color = make_color(tot_color);

	// end gather process

	// draw the tiles & extent

	if (show_gradient_buffer[0] > 0)
	{
		float3 going_to_add_gradient_color = crosshair_buffer[index];
		float3 final_color_make = make_float3(final_color.z, final_color.y, final_color.x)*make_float3(1.0f / 255.99f);

		final_color_make.x += going_to_add_gradient_color.x;
		final_color_make.y += going_to_add_gradient_color.y;
		final_color_make.z += going_to_add_gradient_color.z;

		final_color = make_color(final_color_make);
		output_buffer[index] = final_color;
	}
	else
	{
		output_buffer[index] = final_color;
	}


	// draw the tiles & extent
	if (show_tile_buffer[0] > 0) {
		//if (rand_pixel.x == centre_pixel.x - tilesize / 2u || rand_pixel.y == centre_pixel.y - tilesize / 2u) {
		if (index.x == centre_pixel.x - tilesize / 2u || index.x == centre_pixel.x + tilesize / 2u || index.y == centre_pixel.y - tilesize / 2u || index.y == centre_pixel.y + tilesize / 2u) {
			float3 whiteline = make_float3(255.0f, 255.0f, 255.0f);
			output_buffer[index] = make_color(whiteline);
		}
		else {
			output_buffer[index] = final_color;
		}

	}
	else {
		output_buffer[index] = final_color;
	}
	
}

// implement 3D Gaussian filter using convolution
static __device__ __inline__ void convolution_3D_to_tile_gather()
{
	// record the start time from rendering

	float start_to_reconstruction_time = render_elapse_time_buffer[0];
	clock_t reconstruction_start_time = clock();

	// define the input index as thread index
	uint2 index = make_uint2(launch_index.x, launch_index.y);
	// using the input index to find the tile (tile = grid in this code)
	uint which_grid_calculate = pixel_at_tile_buffer[index];

	// assign a tile if the call is illegal, should not happen after every pixel is updated at least once
	if (which_grid_calculate > 256) {
		which_grid_calculate = launch_index.x;
	}
	// the center pixel of the tile
	uint2 centre_pixel = make_uint2(leaf_tile_indices[which_grid_calculate].x, leaf_tile_indices[which_grid_calculate].y);
	// tile size of the tile
	unsigned int tilesize = leaf_tile_sizes[which_grid_calculate];

	// find the calculated extent(filter size) in this tile
	float3 extents = extent_buffer[index];
	uint3 filter_size = make_uint3((unsigned int)extents.x, (unsigned int)extents.y, (unsigned int)extents.z);


	// reassign the filter size if they are too big

	//float3 going_to_use_this_extent = extent_buffer[index];
	float3 going_to_use_this_extent = extent_at_tile[which_grid_calculate];

	////////////////caution!!!!!!! need to change back if not ok

	//going_to_use_this_extent.x /= 5.5;
	//going_to_use_this_extent.y /= 5.5;
	//going_to_use_this_extent.x -= 2.0;
	//going_to_use_this_extent.y -= 2.0;

	if (launch_index.x == 300u && launch_index.y == 300u) {
		//rtPrintf("which_grid_calculate = %u \n", which_grid_calculate);
		//rtPrintf("filter size x = %f, filter size y = %f, filter size z = %f \n", going_to_use_this_extent.x, going_to_use_this_extent.y, going_to_use_this_extent.z);
	}

	if (going_to_use_this_extent.x < 0)
		going_to_use_this_extent.x = 0.0f;
	if (going_to_use_this_extent.y < 0)
		going_to_use_this_extent.y = 0.0f;

	uint gaussian_filter_size_x = going_to_use_this_extent.x;
	uint gaussian_filter_size_y = going_to_use_this_extent.y;
	uint gaussian_filter_size_z = going_to_use_this_extent.z;


	//gaussian_filter_size_x = 21;
	//gaussian_filter_size_y = 21;

	// reassign the filter size to odd number

	if (gaussian_filter_size_x % 2 == 0u) {
		gaussian_filter_size_x += 1u;
	}

	if (gaussian_filter_size_y % 2 == 0u) {
		gaussian_filter_size_y += 1u;
	}

	if (gaussian_filter_size_z % 2 == 0u) {
		gaussian_filter_size_z += 1u;
	}

	uint temp_filter_size_z = gaussian_filter_size_z;

	// reassign the filter size if they are too big
	if (gaussian_filter_size_x > 33u) {
		gaussian_filter_size_x = 33u;
	}

	if (gaussian_filter_size_y > 33u) {
		gaussian_filter_size_y = 33u;
	}

	//if (gaussian_filter_size_z > 3u) {
	//	gaussian_filter_size_z = 3u;
	//}

	if (temp_filter_size_z > 21u) {
		temp_filter_size_z = 21u;
	}

	if (launch_index.x == 100u && launch_index.y == 100u) {
		//rtPrintf("In recon: x = %d, y = %d, z = %d \n", gaussian_filter_size_x, gaussian_filter_size_y, gaussian_filter_size_z);
		//rtPrintf("which_grid_calculate= %d \n", which_grid_calculate);
	}

	// Test  

	float total_weight = 0.0f;
	uchar4 total_color, final_color;
	float3 tot_color;
	tot_color.x = tot_color.y = tot_color.z = 0.0f;

	// help to find where the start pixel for Gaussian filter
	uint gaussian_start_index_x = launch_index.x - gaussian_filter_size_x / 2 + 1;
	uint gaussian_start_index_y = launch_index.y - gaussian_filter_size_y / 2 + 1;
	uint2 guassian_start_index = make_uint2(gaussian_start_index_x, gaussian_start_index_y);

	// use convolution
	for (unsigned int i = 0u; i < gaussian_filter_size_x; i++)
	{
		for (unsigned int j = 0u; j < gaussian_filter_size_y; j++)
		{
			for (int k = 0; k < 4; k++)
			{
				float Nx, Ny, Nz, N;

				uint color_index_x = guassian_start_index.x + i;
				uint color_index_y = guassian_start_index.y + j;
				uint2 color_index = make_uint2(color_index_x, color_index_y);

				if (color_index_x < 0 || color_index_x > screen_size_buffer[0] || color_index_y < 0 || color_index_y > screen_size_buffer[1]) {
					continue;
				}


				if (gaussian_filter_size_y == 1u) {
					Ny = gaussian_1[j];
				}
				else if (gaussian_filter_size_y == 3u) {
					Ny = gaussian_3[j];
				}
				else if (gaussian_filter_size_y == 5u) {
					Ny = gaussian_5[j];
				}
				else if (gaussian_filter_size_y == 7u) {
					Ny = gaussian_7[j];
				}
				else if (gaussian_filter_size_y == 9u) {
					Ny = gaussian_9[j];
				}
				else if (gaussian_filter_size_y == 11u) {
					Ny = gaussian_11[j];
				}
				else if (gaussian_filter_size_y == 13u) {
					Ny = gaussian_13[j];
				}
				else if (gaussian_filter_size_y == 15u) {
					Ny = gaussian_15[j];
				}
				else if (gaussian_filter_size_y == 17u) {
					Ny = gaussian_17[j];
				}
				else if (gaussian_filter_size_y == 19u) {
					Ny = gaussian_19[j];
				}
				else if (gaussian_filter_size_y == 21u) {
					Ny = gaussian_21[j];
				}
				else if (gaussian_filter_size_y == 23u) {
					Ny = gaussian_23[j];
				}
				else if (gaussian_filter_size_y == 25u) {
					Ny = gaussian_25[j];
				}
				else if (gaussian_filter_size_y == 27u) {
					Ny = gaussian_27[j];
				}
				else if (gaussian_filter_size_y == 29u) {
					Ny = gaussian_29[j];
				}
				else if (gaussian_filter_size_y == 31u) {
					Ny = gaussian_31[j];
				}
				else if (gaussian_filter_size_y == 33u) {
					Ny = gaussian_33[j];
				}
				else {
					Ny = gaussian_33[j];
				}

				if (gaussian_filter_size_x == 1u) {
					Nx = gaussian_1[i];
				}
				else if (gaussian_filter_size_x == 3u) {
					Nx = gaussian_3[i];
				}
				else if (gaussian_filter_size_x == 5u) {
					Nx = gaussian_5[i];
				}
				else if (gaussian_filter_size_x == 7u) {
					Nx = gaussian_7[i];
				}
				else if (gaussian_filter_size_x == 9u) {
					Nx = gaussian_9[i];
				}
				else if (gaussian_filter_size_x == 11u) {
					Nx = gaussian_11[i];
				}
				else if (gaussian_filter_size_x == 13u) {
					Nx = gaussian_13[i];
				}
				else if (gaussian_filter_size_x == 15u) {
					Nx = gaussian_15[i];
				}
				else if (gaussian_filter_size_x == 17u) {
					Nx = gaussian_17[i];
				}
				else if (gaussian_filter_size_x == 19u) {
					Nx = gaussian_19[i];
				}
				else if (gaussian_filter_size_x == 21u) {
					Nx = gaussian_21[i];
				}
				else if (gaussian_filter_size_x == 23u) {
					Nx = gaussian_23[i];
				}
				else if (gaussian_filter_size_x == 25u) {
					Nx = gaussian_25[i];
				}
				else if (gaussian_filter_size_x == 27u) {
					Nx = gaussian_27[i];
				}
				else if (gaussian_filter_size_x == 29u) {
					Nx = gaussian_29[i];
				}
				else if (gaussian_filter_size_x == 31u) {
					Nx = gaussian_31[i];
				}
				else if (gaussian_filter_size_x == 33u) {
					Nx = gaussian_33[i];
				}
				else {
					Nx = gaussian_33[i];
				}

				float sample_age = 0.0f;

				if (k == 0) {
					sample_age = start_to_reconstruction_time - sample_time_buffer[color_index];
				}
				else if (k == 1) {
					sample_age = start_to_reconstruction_time - sample_time_temp_buffer1[color_index];
				}
				else if (k == 2) {
					sample_age = start_to_reconstruction_time - sample_time_temp_buffer2[color_index];
				}
				else if (k == 3) {
					sample_age = start_to_reconstruction_time - sample_time_temp_buffer3[color_index];
				}
				else {
					sample_age = 777.0f;
				}



				float new_temporal_extent = going_to_use_this_extent.z * 2.0;

				//sample_age -= 0.045*k;

				if (sample_age <= 0)
					sample_age = 0;

				//sample_age *= 750;

				//sample_age *= 100;

				uint sample_age_index = (uint)floorf(sample_age);
				uint sample_age_index_ceil = (uint)ceilf(sample_age);

				if (launch_index.x == 50u && launch_index.y == 50u && k == 0) {
					//rtPrintf("sample_age = %f, sample_age_cc = %u sample_age_index = %u\n", sample_age, (uint)ceilf(sample_age), sample_age_index);
					//rtPrintf("extentx = %f, extenty = %f, new temp extentz = %f\n", going_to_use_this_extent.x, going_to_use_this_extent.y, new_temporal_extent);
					//rtPrintf("which_grid_calculate = %d, extent x= %f, extent y = %f, extent z = %f\n", which_grid_calculate, going_to_use_this_extent.x, going_to_use_this_extent.y, going_to_use_this_extent.z);
				}

				// interpolation:
				// A0   : sample_age floor
				// A1   : sample_age ceil
				// A    : current extent (new_temporal_extent)
				// B0   : Gaussian weight for A0
				// B1   : Gaussian weight for A1
				// B(Nz): Expected intepolation weight
				// alpha: (A-A0)/(A1-A0) = A-A0

				float A0, A1, A, B0, B1, alpha;
				A0 = floorf(sample_age);
				A1 = ceilf(sample_age);
				alpha = sample_age - A0;

				//if (launch_index.x == 100u && launch_index.y == 100u && k == 0)
				//	rtPrintf("new_temporal_extent = %f, sample_age = %f\n", new_temporal_extent, sample_age);


				if (sample_age >= 25.0f) {
					Nz = 0.00000;
					//Nz = 0.0000001;
				}
				else if (new_temporal_extent > 49.0f) {
					B0 = gaussianT_49[sample_age_index];
					B1 = gaussianT_49[sample_age_index_ceil];
					Nz = B0 * (1 - alpha) + B1 * alpha;
				}
				else {
					if (new_temporal_extent <= 1u) {
						if (sample_age_index > 0u)
							Nz = gaussianT_1[0];
						else
							Nz = gaussianT_1[sample_age_index];
					}
					else if (new_temporal_extent <= 3u) {

						if (sample_age_index >= 1u)
							Nz = gaussianT_3[1];
						else {
							B0 = gaussianT_3[sample_age_index];
							B1 = gaussianT_3[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 5u) {
						if (sample_age_index >= 2u)
							Nz = gaussianT_5[2];
						else {
							B0 = gaussianT_5[sample_age_index];
							B1 = gaussianT_5[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 7u) {
						if (sample_age_index >= 3u)
							Nz = gaussianT_7[3];
						else {
							B0 = gaussianT_7[sample_age_index];
							B1 = gaussianT_7[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 9u) {
						if (sample_age_index >= 4u)
							Nz = gaussianT_9[4];
						else {
							B0 = gaussianT_9[sample_age_index];
							B1 = gaussianT_9[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 11u) {
						if (sample_age_index >= 5u)
							Nz = gaussianT_11[5];
						else {
							B0 = gaussianT_11[sample_age_index];
							B1 = gaussianT_11[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 13u) {
						if (sample_age_index >= 6u)
							Nz = gaussianT_13[6];
						else {
							B0 = gaussianT_13[sample_age_index];
							B1 = gaussianT_13[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 15u) {
						if (sample_age_index >= 7u)
							Nz = gaussianT_15[7];
						else {
							B0 = gaussianT_15[sample_age_index];
							B1 = gaussianT_15[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 17u) {
						if (sample_age_index >= 8u)
							Nz = gaussianT_17[8];
						else {
							B0 = gaussianT_17[sample_age_index];
							B1 = gaussianT_17[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 19u) {
						if (sample_age_index >= 9u)
							Nz = gaussianT_19[9];
						else {
							B0 = gaussianT_19[sample_age_index];
							B1 = gaussianT_19[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 21u) {
						if (sample_age_index >= 10u)
							Nz = gaussianT_21[10];
						else {
							B0 = gaussianT_21[sample_age_index];
							B1 = gaussianT_21[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 23u) {
						if (sample_age_index >= 11u)
							Nz = gaussianT_23[11];
						else {
							B0 = gaussianT_23[sample_age_index];
							B1 = gaussianT_23[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 25u) {
						if (sample_age_index >= 12u)
							Nz = gaussianT_25[12];
						else {
							B0 = gaussianT_25[sample_age_index];
							B1 = gaussianT_25[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 27u) {
						if (sample_age_index >= 13u)
							Nz = gaussianT_27[13];
						else {
							B0 = gaussianT_27[sample_age_index];
							B1 = gaussianT_27[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 29u) {
						if (sample_age_index >= 14u)
							Nz = gaussianT_29[14];
						else {
							B0 = gaussianT_29[sample_age_index];
							B1 = gaussianT_29[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 31u) {
						if (sample_age_index >= 15u)
							Nz = gaussianT_31[15];
						else {
							B0 = gaussianT_31[sample_age_index];
							B1 = gaussianT_31[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 33u) {
						if (sample_age_index >= 16u)
							Nz = gaussianT_33[16];
						else {
							B0 = gaussianT_33[sample_age_index];
							B1 = gaussianT_33[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 35u) {
						if (sample_age_index >= 17u)
							Nz = gaussianT_35[17];
						else {
							B0 = gaussianT_35[sample_age_index];
							B1 = gaussianT_35[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 37u) {
						if (sample_age_index >= 18u)
							Nz = gaussianT_37[18];
						else {
							B0 = gaussianT_37[sample_age_index];
							B1 = gaussianT_37[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 39u) {
						if (sample_age_index >= 19u)
							Nz = gaussianT_39[19];
						else {
							B0 = gaussianT_39[sample_age_index];
							B1 = gaussianT_39[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 41u) {
						if (sample_age_index >= 20u)
							Nz = gaussianT_41[20];
						else {
							B0 = gaussianT_41[sample_age_index];
							B1 = gaussianT_41[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 43u) {
						if (sample_age_index >= 21u)
							Nz = gaussianT_43[21];
						else {
							B0 = gaussianT_43[sample_age_index];
							B1 = gaussianT_43[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 45u) {
						if (sample_age_index >= 22u)
							Nz = gaussianT_45[22];
						else {
							B0 = gaussianT_45[sample_age_index];
							B1 = gaussianT_45[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 47u) {
						if (sample_age_index >= 23u)
							Nz = gaussianT_47[23];
						else {
							B0 = gaussianT_47[sample_age_index];
							B1 = gaussianT_47[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else {
						if (sample_age_index >= 24u)
							Nz = gaussianT_49[24];
						else {
							B0 = gaussianT_49[sample_age_index];
							B1 = gaussianT_49[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
				}

				if (Nz == 0.0f) {
					Nz = 0.000001;
				}

				/*
				if (gaussian_filter_size_z == 1u) {
				Nz = gaussian_1[k];
				}
				else if (gaussian_filter_size_z == 3u) {
				Nz = gaussian_3[k];
				}


				if (temp_filter_size_z == 1u) {
				Nz = gaussian_1[k];
				}
				else if (temp_filter_size_z == 3u) {
				Nz = gaussian_3[k];
				}
				else if (temp_filter_size_z == 5u) {
				Nz = gaussian_5[k];
				}
				else if (temp_filter_size_z == 7u) {
				Nz = gaussian_7[k];
				}
				else if (temp_filter_size_z == 9u) {
				Nz = gaussian_9[k];
				}
				else if (temp_filter_size_z == 11u) {
				Nz = gaussian_11[k];
				}
				else if (temp_filter_size_z == 13u) {
				Nz = gaussian_13[k];
				}
				else if (temp_filter_size_z == 15u) {
				Nz = gaussian_15[k];
				}
				else if (temp_filter_size_z == 17u) {
				Nz = gaussian_17[k];
				}
				else if (temp_filter_size_z == 19u) {
				Nz = gaussian_19[k];
				}
				else {
				Nz = gaussian_21[k];
				}
				*/


				N = Nx * Ny*Nz;


				uchar4 color_info = color_buffer[color_index];

				if (k == 0) {
					color_info = color_buffer[color_index];
				}
				else if (k == 1) {
					color_info = temp_buffer1[color_index];
				}
				else if (k == 2) {
					color_info = temp_buffer2[color_index];
				}
				else {
					color_info = temp_buffer3[color_index];
				}



				float3 color_info_float = make_float3(color_info.z, color_info.y, color_info.x)*make_float3(1.0f / 255.99f);

				color_info_float *= make_float3(N);
				tot_color += color_info_float;
				total_weight += N;
			}
		}
	}//for loop ends
	if (total_weight != 0) {
		tot_color *= make_float3(1.0f / total_weight);
		final_color = make_color(tot_color);
	}
	else {
		//tot_color.x = 255.0f;
		//tot_color.y = tot_color.z = 0.0f;
		//final_color = make_color(tot_color);
	}

	if (launch_index.x == 128u && launch_index.y == 128u) {
		//rtPrintf("In recon: x = %d, y = %d, z = %d t = %f\n", gaussian_filter_size_x, gaussian_filter_size_y, gaussian_filter_size_z, going_to_use_this_extent.z);
		//rtPrintf("which_grid_calculate= %d \n", which_grid_calculate);
	}

	// end gather process
	// draw the tiles & extent

	if (show_gradient_buffer[0] > 0)
	{
		float3 going_to_add_gradient_color = crosshair_buffer[index];

		//going_to_add_gradient_color = 

		float3 final_color_make = make_float3(final_color.z, final_color.y, final_color.x)*make_float3(1.0f / 255.99f);

		final_color_make.x = going_to_add_gradient_color.x;
		final_color_make.y = going_to_add_gradient_color.y;
		final_color_make.z = going_to_add_gradient_color.z;
		float g_x = (float)gaussian_filter_size_x - 1;
		float g_y = (float)gaussian_filter_size_y - 1;
		//if (launch_index.x == 128u && launch_index.y == 128u)
		//	rtPrintf("g_x = %f\n", g_x);
		//final_color_make.x += (g_x * (1.0f / 255.99f)*5) ;
		//final_color_make.y += (g_y * (1.0f / 255.99f)*5);
		//final_color_make.z += 0.0;

		final_color = make_color(final_color_make);
		output_buffer[index] = final_color;
	}
	else
	{
		output_buffer[index] = final_color;
	}


	if (show_tile_buffer[0] > 0) {
		//if (rand_pixel.x == centre_pixel.x - tilesize / 2u || rand_pixel.y == centre_pixel.y - tilesize / 2u) {
		if (index.x == centre_pixel.x - tilesize / 2u || index.x == centre_pixel.x + tilesize / 2u || index.y == centre_pixel.y - tilesize / 2u || index.y == centre_pixel.y + tilesize / 2u) {
			float3 whiteline = make_float3(255.0f, 255.0f, 255.0f);
			output_buffer[index] = make_color(whiteline);
		}
		else {
			output_buffer[index] = final_color;
		}

	}
	else {
		output_buffer[index] = final_color;
	}
}


// implement 3D Gaussian filter using convolution
static __device__ __inline__ void convolution_3D_to_tile_gather_copy()
{
	// record the start time from rendering

	float start_to_reconstruction_time = render_elapse_time_buffer[0];
	clock_t reconstruction_start_time = clock();

	// define the input index as thread index
	uint2 index = make_uint2(launch_index.x, launch_index.y);
	// using the input index to find the tile (tile = grid in this code)
	uint which_grid_calculate = pixel_at_tile_buffer[index];

	// assign a tile if the call is illegal, should not happen after every pixel is updated at least once
	if (which_grid_calculate > 512) {
		which_grid_calculate = launch_index.x;
	}
	// the center pixel of the tile
	uint2 centre_pixel = make_uint2(leaf_tile_indices[which_grid_calculate].x, leaf_tile_indices[which_grid_calculate].y);
	// tile size of the tile
	unsigned int tilesize = leaf_tile_sizes[which_grid_calculate];

	// find the calculated extent(filter size) in this tile
	float3 extents = extent_buffer[index];
	uint3 filter_size = make_uint3((unsigned int)extents.x, (unsigned int)extents.y, (unsigned int)extents.z);


	// reassign the filter size if they are too big

	//float3 going_to_use_this_extent = extent_buffer[index];
	float3 going_to_use_this_extent = extent_at_tile[which_grid_calculate];

	////////////////caution!!!!!!! need to change back if not ok

	//going_to_use_this_extent.x *= 1.5;
	//going_to_use_this_extent.y *= 1.5;
	//going_to_use_this_extent.x -= 2.0;
	//going_to_use_this_extent.y -= 2.0;

	if (launch_index.x == 300u && launch_index.y == 300u) {
		//rtPrintf("which_grid_calculate = %u \n", which_grid_calculate);
		//rtPrintf("filter size x = %f, filter size y = %f, filter size z = %f \n", going_to_use_this_extent.x, going_to_use_this_extent.y, going_to_use_this_extent.z);
	}

	if (going_to_use_this_extent.x < 0)
		going_to_use_this_extent.x = 0.0f;
	if (going_to_use_this_extent.y < 0)
		going_to_use_this_extent.y = 0.0f;

	uint gaussian_filter_size_x = going_to_use_this_extent.x;
	uint gaussian_filter_size_y = going_to_use_this_extent.y;
	uint gaussian_filter_size_z = going_to_use_this_extent.z;


	//gaussian_filter_size_x = 21;
	//gaussian_filter_size_y = 21;

	// reassign the filter size to odd number

	if (gaussian_filter_size_x % 2 == 0u) {
		gaussian_filter_size_x += 1u;
	}

	if (gaussian_filter_size_y % 2 == 0u) {
		gaussian_filter_size_y += 1u;
	}

	if (gaussian_filter_size_z % 2 == 0u) {
		gaussian_filter_size_z += 1u;
	}

	uint temp_filter_size_z = gaussian_filter_size_z;

	// reassign the filter size if they are too big
	if (gaussian_filter_size_x > 33u) {
		gaussian_filter_size_x = 33u;
	}

	if (gaussian_filter_size_y > 33u) {
		gaussian_filter_size_y = 33u;
	}

	//if (gaussian_filter_size_z > 3u) {
	//	gaussian_filter_size_z = 3u;
	//}

	if (temp_filter_size_z > 21u) {
		temp_filter_size_z = 21u;
	}

	if (launch_index.x == 100u && launch_index.y == 100u) {
		//rtPrintf("In recon: x = %d, y = %d, z = %d \n", gaussian_filter_size_x, gaussian_filter_size_y, gaussian_filter_size_z);
		//rtPrintf("which_grid_calculate= %d \n", which_grid_calculate);
	}

	// Test  

	float total_weight = 0.0f;
	uchar4 total_color, final_color;
	float3 tot_color;
	tot_color.x = tot_color.y = tot_color.z = 0.0f;

	if (is_moving_buffer[0] == 0) {
		gaussian_filter_size_x = 1;
		gaussian_filter_size_y = 1;
	}


	// help to find where the start pixel for Gaussian filter
	uint gaussian_start_index_x = launch_index.x - gaussian_filter_size_x / 2 + 1;
	uint gaussian_start_index_y = launch_index.y - gaussian_filter_size_y / 2 + 1;
	uint2 guassian_start_index = make_uint2(gaussian_start_index_x, gaussian_start_index_y);

	// use convolution
	for (unsigned int i = 0u; i < gaussian_filter_size_x; i++)
	{
		for (unsigned int j = 0u; j < gaussian_filter_size_y; j++)
		{
			for (int k = 0; k < 4; k++)
			{
				float Nx, Ny, Nz, N;

				uint color_index_x = guassian_start_index.x + i;
				uint color_index_y = guassian_start_index.y + j;
				uint2 color_index = make_uint2(color_index_x, color_index_y);

				if (color_index_x < 0 || color_index_x > screen_size_buffer[0] || color_index_y < 0 || color_index_y > screen_size_buffer[1]) {
					continue;
				}


				if (gaussian_filter_size_y == 1u) {
					Ny = gaussian_1[j];
				}
				else if (gaussian_filter_size_y == 3u) {
					Ny = gaussian_3[j];
				}
				else if (gaussian_filter_size_y == 5u) {
					Ny = gaussian_5[j];
				}
				else if (gaussian_filter_size_y == 7u) {
					Ny = gaussian_7[j];
				}
				else if (gaussian_filter_size_y == 9u) {
					Ny = gaussian_9[j];
				}
				else if (gaussian_filter_size_y == 11u) {
					Ny = gaussian_11[j];
				}
				else if (gaussian_filter_size_y == 13u) {
					Ny = gaussian_13[j];
				}
				else if (gaussian_filter_size_y == 15u) {
					Ny = gaussian_15[j];
				}
				else if (gaussian_filter_size_y == 17u) {
					Ny = gaussian_17[j];
				}
				else if (gaussian_filter_size_y == 19u) {
					Ny = gaussian_19[j];
				}
				else if (gaussian_filter_size_y == 21u) {
					Ny = gaussian_21[j];
				}
				else if (gaussian_filter_size_y == 23u) {
					Ny = gaussian_23[j];
				}
				else if (gaussian_filter_size_y == 25u) {
					Ny = gaussian_25[j];
				}
				else if (gaussian_filter_size_y == 27u) {
					Ny = gaussian_27[j];
				}
				else if (gaussian_filter_size_y == 29u) {
					Ny = gaussian_29[j];
				}
				else if (gaussian_filter_size_y == 31u) {
					Ny = gaussian_31[j];
				}
				else if (gaussian_filter_size_y == 33u) {
					Ny = gaussian_33[j];
				}
				else {
					Ny = gaussian_33[j];
				}

				if (gaussian_filter_size_x == 1u) {
					Nx = gaussian_1[i];
				}
				else if (gaussian_filter_size_x == 3u) {
					Nx = gaussian_3[i];
				}
				else if (gaussian_filter_size_x == 5u) {
					Nx = gaussian_5[i];
				}
				else if (gaussian_filter_size_x == 7u) {
					Nx = gaussian_7[i];
				}
				else if (gaussian_filter_size_x == 9u) {
					Nx = gaussian_9[i];
				}
				else if (gaussian_filter_size_x == 11u) {
					Nx = gaussian_11[i];
				}
				else if (gaussian_filter_size_x == 13u) {
					Nx = gaussian_13[i];
				}
				else if (gaussian_filter_size_x == 15u) {
					Nx = gaussian_15[i];
				}
				else if (gaussian_filter_size_x == 17u) {
					Nx = gaussian_17[i];
				}
				else if (gaussian_filter_size_x == 19u) {
					Nx = gaussian_19[i];
				}
				else if (gaussian_filter_size_x == 21u) {
					Nx = gaussian_21[i];
				}
				else if (gaussian_filter_size_x == 23u) {
					Nx = gaussian_23[i];
				}
				else if (gaussian_filter_size_x == 25u) {
					Nx = gaussian_25[i];
				}
				else if (gaussian_filter_size_x == 27u) {
					Nx = gaussian_27[i];
				}
				else if (gaussian_filter_size_x == 29u) {
					Nx = gaussian_29[i];
				}
				else if (gaussian_filter_size_x == 31u) {
					Nx = gaussian_31[i];
				}
				else if (gaussian_filter_size_x == 33u) {
					Nx = gaussian_33[i];
				}
				else {
					Nx = gaussian_33[i];
				}

				float sample_age = 0.0f;

				if (k == 0) {
					sample_age = start_to_reconstruction_time - sample_time_buffer[color_index];
				}
				else if (k == 1) {
					sample_age = start_to_reconstruction_time - sample_time_temp_buffer1[color_index];
				}
				else if (k == 2) {
					sample_age = start_to_reconstruction_time - sample_time_temp_buffer2[color_index];
				}
				else if (k == 3) {
					sample_age = start_to_reconstruction_time - sample_time_temp_buffer3[color_index];
				}
				else {
					sample_age = 777.0f;
				}



				float new_temporal_extent = going_to_use_this_extent.z * 2.0;

				//sample_age -= 0.045*k;

				if (sample_age <= 0)
					sample_age = 0;

				//sample_age *= 750;

				//sample_age *= 100;

				uint sample_age_index = (uint)floorf(sample_age);
				uint sample_age_index_ceil = (uint)ceilf(sample_age);

				if (launch_index.x == 50u && launch_index.y == 50u && k == 0) {
					//rtPrintf("sample_age = %f, sample_age_cc = %u sample_age_index = %u\n", sample_age, (uint)ceilf(sample_age), sample_age_index);
					//rtPrintf("extentx = %f, extenty = %f, new temp extentz = %f\n", going_to_use_this_extent.x, going_to_use_this_extent.y, new_temporal_extent);
					//rtPrintf("which_grid_calculate = %d, extent x= %f, extent y = %f, extent z = %f\n", which_grid_calculate, going_to_use_this_extent.x, going_to_use_this_extent.y, going_to_use_this_extent.z);
				}

				// interpolation:
				// A0   : sample_age floor
				// A1   : sample_age ceil
				// A    : current extent (new_temporal_extent)
				// B0   : Gaussian weight for A0
				// B1   : Gaussian weight for A1
				// B(Nz): Expected intepolation weight
				// alpha: (A-A0)/(A1-A0) = A-A0

				float A0, A1, A, B0, B1, alpha;
				A0 = floorf(sample_age);
				A1 = ceilf(sample_age);
				alpha = sample_age - A0;

				//if (launch_index.x == 100u && launch_index.y == 100u && k == 0)
				//	rtPrintf("new_temporal_extent = %f, sample_age = %f\n", new_temporal_extent, sample_age);


				if (sample_age >= 25.0f) {
					Nz = 0.00000;
					//Nz = 0.0000001;
				}
				else if (new_temporal_extent > 49.0f) {
					B0 = gaussianT_49[sample_age_index];
					B1 = gaussianT_49[sample_age_index_ceil];
					Nz = B0 * (1 - alpha) + B1 * alpha;
				}
				else {
					if (new_temporal_extent <= 1u) {
						if (sample_age_index > 0u)
							Nz = gaussianT_1[0];
						else
							Nz = gaussianT_1[sample_age_index];
					}
					else if (new_temporal_extent <= 3u) {

						if (sample_age_index >= 1u)
							Nz = gaussianT_3[1];
						else {
							B0 = gaussianT_3[sample_age_index];
							B1 = gaussianT_3[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 5u) {
						if (sample_age_index >= 2u)
							Nz = gaussianT_5[2];
						else {
							B0 = gaussianT_5[sample_age_index];
							B1 = gaussianT_5[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 7u) {
						if (sample_age_index >= 3u)
							Nz = gaussianT_7[3];
						else {
							B0 = gaussianT_7[sample_age_index];
							B1 = gaussianT_7[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 9u) {
						if (sample_age_index >= 4u)
							Nz = gaussianT_9[4];
						else {
							B0 = gaussianT_9[sample_age_index];
							B1 = gaussianT_9[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 11u) {
						if (sample_age_index >= 5u)
							Nz = gaussianT_11[5];
						else {
							B0 = gaussianT_11[sample_age_index];
							B1 = gaussianT_11[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 13u) {
						if (sample_age_index >= 6u)
							Nz = gaussianT_13[6];
						else {
							B0 = gaussianT_13[sample_age_index];
							B1 = gaussianT_13[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 15u) {
						if (sample_age_index >= 7u)
							Nz = gaussianT_15[7];
						else {
							B0 = gaussianT_15[sample_age_index];
							B1 = gaussianT_15[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 17u) {
						if (sample_age_index >= 8u)
							Nz = gaussianT_17[8];
						else {
							B0 = gaussianT_17[sample_age_index];
							B1 = gaussianT_17[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 19u) {
						if (sample_age_index >= 9u)
							Nz = gaussianT_19[9];
						else {
							B0 = gaussianT_19[sample_age_index];
							B1 = gaussianT_19[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 21u) {
						if (sample_age_index >= 10u)
							Nz = gaussianT_21[10];
						else {
							B0 = gaussianT_21[sample_age_index];
							B1 = gaussianT_21[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 23u) {
						if (sample_age_index >= 11u)
							Nz = gaussianT_23[11];
						else {
							B0 = gaussianT_23[sample_age_index];
							B1 = gaussianT_23[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 25u) {
						if (sample_age_index >= 12u)
							Nz = gaussianT_25[12];
						else {
							B0 = gaussianT_25[sample_age_index];
							B1 = gaussianT_25[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 27u) {
						if (sample_age_index >= 13u)
							Nz = gaussianT_27[13];
						else {
							B0 = gaussianT_27[sample_age_index];
							B1 = gaussianT_27[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 29u) {
						if (sample_age_index >= 14u)
							Nz = gaussianT_29[14];
						else {
							B0 = gaussianT_29[sample_age_index];
							B1 = gaussianT_29[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 31u) {
						if (sample_age_index >= 15u)
							Nz = gaussianT_31[15];
						else {
							B0 = gaussianT_31[sample_age_index];
							B1 = gaussianT_31[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 33u) {
						if (sample_age_index >= 16u)
							Nz = gaussianT_33[16];
						else {
							B0 = gaussianT_33[sample_age_index];
							B1 = gaussianT_33[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 35u) {
						if (sample_age_index >= 17u)
							Nz = gaussianT_35[17];
						else {
							B0 = gaussianT_35[sample_age_index];
							B1 = gaussianT_35[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 37u) {
						if (sample_age_index >= 18u)
							Nz = gaussianT_37[18];
						else {
							B0 = gaussianT_37[sample_age_index];
							B1 = gaussianT_37[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 39u) {
						if (sample_age_index >= 19u)
							Nz = gaussianT_39[19];
						else {
							B0 = gaussianT_39[sample_age_index];
							B1 = gaussianT_39[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 41u) {
						if (sample_age_index >= 20u)
							Nz = gaussianT_41[20];
						else {
							B0 = gaussianT_41[sample_age_index];
							B1 = gaussianT_41[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 43u) {
						if (sample_age_index >= 21u)
							Nz = gaussianT_43[21];
						else {
							B0 = gaussianT_43[sample_age_index];
							B1 = gaussianT_43[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 45u) {
						if (sample_age_index >= 22u)
							Nz = gaussianT_45[22];
						else {
							B0 = gaussianT_45[sample_age_index];
							B1 = gaussianT_45[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else if (new_temporal_extent <= 47u) {
						if (sample_age_index >= 23u)
							Nz = gaussianT_47[23];
						else {
							B0 = gaussianT_47[sample_age_index];
							B1 = gaussianT_47[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
					else {
						if (sample_age_index >= 24u)
							Nz = gaussianT_49[24];
						else {
							B0 = gaussianT_49[sample_age_index];
							B1 = gaussianT_49[sample_age_index_ceil];
							Nz = B0 * (1 - alpha) + B1 * alpha;
						}
					}
				}

				if (Nz == 0.0f) {
					Nz = 0.000001;
				}

				/*
				if (gaussian_filter_size_z == 1u) {
				Nz = gaussian_1[k];
				}
				else if (gaussian_filter_size_z == 3u) {
				Nz = gaussian_3[k];
				}


				if (temp_filter_size_z == 1u) {
				Nz = gaussian_1[k];
				}
				else if (temp_filter_size_z == 3u) {
				Nz = gaussian_3[k];
				}
				else if (temp_filter_size_z == 5u) {
				Nz = gaussian_5[k];
				}
				else if (temp_filter_size_z == 7u) {
				Nz = gaussian_7[k];
				}
				else if (temp_filter_size_z == 9u) {
				Nz = gaussian_9[k];
				}
				else if (temp_filter_size_z == 11u) {
				Nz = gaussian_11[k];
				}
				else if (temp_filter_size_z == 13u) {
				Nz = gaussian_13[k];
				}
				else if (temp_filter_size_z == 15u) {
				Nz = gaussian_15[k];
				}
				else if (temp_filter_size_z == 17u) {
				Nz = gaussian_17[k];
				}
				else if (temp_filter_size_z == 19u) {
				Nz = gaussian_19[k];
				}
				else {
				Nz = gaussian_21[k];
				}
				*/


				N = Nx * Ny*Nz;


				uchar4 color_info = color_buffer[color_index];

				if (k == 0) {
					color_info = color_buffer[color_index];
				}
				else if (k == 1) {
					color_info = temp_buffer1[color_index];
				}
				else if (k == 2) {
					color_info = temp_buffer2[color_index];
				}
				else {
					color_info = temp_buffer3[color_index];
				}



				float3 color_info_float = make_float3(color_info.z, color_info.y, color_info.x)*make_float3(1.0f / 255.99f);

				color_info_float *= make_float3(N);
				tot_color += color_info_float;
				total_weight += N;
			}
		}
	}//for loop ends

	// averaging all the old samples
	if (is_moving_buffer[0] == 0) {
	
		tot_color.x = tot_color.y = tot_color.z = 0.0f;
		uchar4 color_info = color_buffer[launch_index];
		float3 color_info_float = make_float3(color_info.z, color_info.y, color_info.x)*make_float3(1.0f / 255.99f);

		uchar4 color_info1 = color_buffer[launch_index];
		float3 color_info_float1 = make_float3(color_info1.z, color_info1.y, color_info1.x)*make_float3(1.0f / 255.99f);

		uchar4 color_info2 = color_buffer[launch_index];
		float3 color_info_float2 = make_float3(color_info2.z, color_info2.y, color_info2.x)*make_float3(1.0f / 255.99f);

		uchar4 color_info3 = color_buffer[launch_index];
		float3 color_info_float3 = make_float3(color_info3.z, color_info3.y, color_info3.x)*make_float3(1.0f / 255.99f);

		tot_color = color_info_float + color_info_float1 + color_info_float2 + color_info_float3;
		tot_color /= 4;
		total_weight = 1.0f;
	}



	if (total_weight != 0) {
		tot_color *= make_float3(1.0f / total_weight);
		final_color = make_color(tot_color);
	}
	else {
		//tot_color.x = 255.0f;
		//tot_color.y = tot_color.z = 0.0f;
		//final_color = make_color(tot_color);
	}

	if (launch_index.x == 128u && launch_index.y == 128u) {
		//rtPrintf("In recon: x = %d, y = %d, z = %d t = %f\n", gaussian_filter_size_x, gaussian_filter_size_y, gaussian_filter_size_z, going_to_use_this_extent.z);
		//rtPrintf("which_grid_calculate= %d \n", which_grid_calculate);
	}

	// end gather process
	// draw the tiles & extent

	if (show_gradient_buffer[0] > 0)
	{
		float3 going_to_add_gradient_color = crosshair_buffer[index];

		//going_to_add_gradient_color = 

		float3 final_color_make = make_float3(final_color.z, final_color.y, final_color.x)*make_float3(1.0f / 255.99f);

		final_color_make.x = going_to_add_gradient_color.x;
		final_color_make.y = going_to_add_gradient_color.y;
		final_color_make.z = going_to_add_gradient_color.z;
		float g_x = (float)gaussian_filter_size_x - 1;
		float g_y = (float)gaussian_filter_size_y - 1;
		//if (launch_index.x == 128u && launch_index.y == 128u)
		//	rtPrintf("g_x = %f\n", g_x);
		//final_color_make.x += (g_x * (1.0f / 255.99f)*5) ;
		//final_color_make.y += (g_y * (1.0f / 255.99f)*5);
		//final_color_make.z += 0.0;

		final_color = make_color(final_color_make);
		output_buffer[index] = final_color;
	}
	else
	{
		output_buffer[index] = final_color;
	}


	if (show_tile_buffer[0] > 0) {
		//if (rand_pixel.x == centre_pixel.x - tilesize / 2u || rand_pixel.y == centre_pixel.y - tilesize / 2u) {
		if (index.x == centre_pixel.x - tilesize / 2u || index.x == centre_pixel.x + tilesize / 2u || index.y == centre_pixel.y - tilesize / 2u || index.y == centre_pixel.y + tilesize / 2u) {
			float3 whiteline = make_float3(255.0f, 255.0f, 255.0f);
			output_buffer[index] = make_color(whiteline);
		}
		else {
			output_buffer[index] = final_color;
		}

	}
	else {
		output_buffer[index] = final_color;
	}
}



RT_PROGRAM void pinhole_camera()
{
	/*
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon );

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_color( prd.result );
	*/

	//frameless_rendering();
	//shoot_3rays_use_random_map_tile();
	
	shoot_3rays_new(); 
	//shoot_3rays_use_random_map_tile_test();

}

RT_PROGRAM void extent_calculation() {
	if (launch_index.y <= 0u)
	{
		new_calculate_variance(true);
	}
	else if (launch_index.y <= 1u && launch_index.x <= (unsigned int)number_of_parent_tiles)
	{
		new_calculate_variance(false);
	}
	else {
		calculate_extent();
	}
}

RT_PROGRAM void reconstruct() 
{
	//frameless_rendering();
	//no_reconstruction();
	//gaussian_filter_to_whole_image_gather();
	
	//convolution_3D_to_tile_gather();
	convolution_3D_to_tile_gather_copy();
}

//
// (NEW)
// Environment map background
//
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
  float theta = atan2f( ray.direction.x, ray.direction.z );
  float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
  float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v     = 0.5f * ( 1.0f + sin(phi) );
  prd_radiance.result = make_float3( tex2D(envmap, u, v) );
}
  

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);

  rtTerminateRay();
}
  

//
// Phong surface shading with shadows 
//
rtDeclareVariable(float3,   Kd, , );
rtDeclareVariable(float3,   Ka, , );
rtDeclareVariable(float3,   Ks, , );
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(float3,   ambient_light_color, , );
rtBuffer<BasicLight>        lights;
rtDeclareVariable(rtObject, top_shadower, , );

RT_PROGRAM void closest_hit_radiance3()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);

    if( nDl > 0.0f ){
      // cast shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      float Ldist = length(light.pos - hit_point);
      optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      float3 light_attenuation = shadow_prd.attenuation;

      if( fmaxf(light_attenuation) > 0.0f ){
        float3 Lc = light.color * light_attenuation;
        color += Kd * nDl * Lc;

        float3 H = normalize(L - ray.direction);
        float nDh = dot( ffnormal, H );
        if(nDh > 0)
          color += Ks * Lc * pow(nDh, phong_exp);
      }

    }
  }
  prd_radiance.result = color;
}


//
// Phong surface shading with shadows and reflections
//
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

RT_PROGRAM void floor_closest_hit_radiance4()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;

  for(int i = 0; i < lights.size(); ++i) {
    BasicLight light = lights[i];
    float3 L = normalize(light.pos - hit_point);
    float nDl = dot( ffnormal, L);

    if( nDl > 0.0f ){
      // cast shadow ray
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      float Ldist = length(light.pos - hit_point);
      optix::Ray shadow_ray( hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      float3 light_attenuation = shadow_prd.attenuation;

      if( fmaxf(light_attenuation) > 0.0f ){
        float3 Lc = light.color * light_attenuation;
        color += Kd * nDl * Lc;

        float3 H = normalize(L - ray.direction);
        float nDh = dot( ffnormal, H );
        if(nDh > 0)
          color += Ks * Lc * pow(nDh, phong_exp);
      }

    }
  }

  float importance = prd_radiance.importance * optix::luminance( reflectivity );

  // reflection ray
  if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
    PerRayData_radiance refl_prd;
    refl_prd.importance = importance;
    refl_prd.depth = prd_radiance.depth+1;
    float3 R = reflect( ray.direction, ffnormal );
    optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
    rtTrace(top_object, refl_ray, refl_prd);
    color += reflectivity * refl_prd.result;
  }

  prd_radiance.result = color;
}
 
// adding some chull implementation from other tutorial to add refraction

//
// Bounding box program for programmable convex hull primitive
//
rtDeclareVariable(float3, chull_bbmin, , );
rtDeclareVariable(float3, chull_bbmax, , );

RT_PROGRAM void chull_bounds(int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->m_min = chull_bbmin;
	aabb->m_max = chull_bbmax;
}

//
// Intersection program for programmable convex hull primitive
//
rtBuffer<float4> planes;
RT_PROGRAM void chull_intersect(int primIdx)
{
	int n = planes.size();
	float t0 = -FLT_MAX;
	float t1 = FLT_MAX;
	float3 t0_normal = make_float3(0);
	float3 t1_normal = make_float3(0);
	for (int i = 0; i < n && t0 < t1; ++i) {
		float4 plane = planes[i];
		float3 n = make_float3(plane);
		float  d = plane.w;

		float denom = dot(n, ray.direction);
		float t = -(d + dot(n, ray.origin)) / denom;
		if (denom < 0) {
			// enter
			if (t > t0) {
				t0 = t;
				t0_normal = n;
			}
		}
		else {
			//exit
			if (t < t1) {
				t1 = t;
				t1_normal = n;
			}
		}
	}

	if (t0 > t1)
		return;

	if (rtPotentialIntersection(t0)) {
		shading_normal = geometric_normal = t0_normal;
		rtReportIntersection(0);
	}
	else if (rtPotentialIntersection(t1)) {
		shading_normal = geometric_normal = t1_normal;
		rtReportIntersection(0);
	}
}

//
// Attenuates shadow rays for shadowing transparent objects
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void glass_any_hit_shadow()
{
	float3 world_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float nDi = fabs(dot(world_normal, ray.direction));

	prd_shadow.attenuation *= 1 - fresnel_schlick(nDi, 5, 1 - shadow_attenuation, make_float3(1));

	rtIgnoreIntersection();
}


//
// Dielectric surface shader
//
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(int, refraction_maxdepth, , );
rtDeclareVariable(int, reflection_maxdepth, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );
RT_PROGRAM void glass_closest_hit_radiance()
{
	// intersection vectors
	const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;                                            // incident direction

	float reflection = 1.0f;
	float3 result = make_float3(0.0f);

	float3 beer_attenuation;
	if (dot(n, ray.direction) > 0) {
		// Beer's law attenuation
		beer_attenuation = exp(extinction_constant * t_hit);
	}
	else {
		beer_attenuation = make_float3(1);
	}

	// refraction
	if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
	{
		float3 t;                                                            // transmission direction
		if (refract(t, i, n, refraction_index))
		{

			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if (cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, n);

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if (importance > importance_cutoff) {
				optix::Ray ray(h, t, radiance_ray_type, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth + 1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += (1.0f - reflection) * refraction_color * refr_prd.result;
			}
			else {
				result += (1.0f - reflection) * refraction_color * cutoff_color;
			}
		}
		// else TIR
	}

	// reflection
	if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
	{
		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if (importance > importance_cutoff) {
			optix::Ray ray(h, r, radiance_ray_type, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 1;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += reflection * reflection_color * refl_prd.result;
		}
		else {
			result += reflection * reflection_color * cutoff_color;
		}
	}

	result = result * beer_attenuation;

	prd_radiance.result = result;
}

//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
